#include "hip/hip_runtime.h"
// Copyright (C) 2023-2025 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <micm/cuda/util/cuda_param.hpp>
#include <micm/cuda/util/cuda_util.cuh>

#include <chrono>

namespace micm
{
  namespace cuda
  {
    /// This is the CUDA kernel that performs the "solve" function on the device
    __global__ void
    SolveKernel(CudaMatrixParam x_param, const CudaMatrixParam ALU_param, const LinearSolverInPlaceParam devstruct)
    {
      // Calculate global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;

      // Local device variables
      const std::size_t* const __restrict__ d_nLij = devstruct.nLij_;
      const std::pair<std::size_t, std::size_t>* __restrict__ d_Lij_yj = devstruct.Lij_yj_;
      const std::pair<std::size_t, std::size_t>* const __restrict__ d_nUij_Uii = devstruct.nUij_Uii_;
      const std::pair<std::size_t, std::size_t>* __restrict__ d_Uij_xj = devstruct.Uij_xj_;
      const std::size_t d_nLij_size = devstruct.nLij_size_;
      const std::size_t d_nUij_Uii_size = devstruct.nUij_Uii_size_;

      const double* const __restrict__ d_ALU = ALU_param.d_data_;
      double* const d_x = x_param.d_data_;
      double* d_y = d_x;  // Alias d_x for consistency with equation, but to reuse memory
      const std::size_t number_of_grid_cells = x_param.number_of_grid_cells_;
      const std::size_t number_of_elements = x_param.number_of_elements_;

      if (tid < number_of_grid_cells)
      {
        // Forward Substitution
        {
          for (auto i = 0; i < d_nLij_size; ++i)
          {
            const std::size_t j_lim = d_nLij[i];
            for (auto j = 0; j < j_lim; ++j)
            {
              const std::size_t d_Lij_yj_first = (*d_Lij_yj).first;
              const std::size_t d_Lij_yj_second_times_ncells = (*d_Lij_yj).second * number_of_grid_cells;
              auto d_ALU_ptr = d_ALU + d_Lij_yj_first;
              auto d_x_ptr = d_x + d_Lij_yj_second_times_ncells;
              d_y[tid] -= d_ALU_ptr[tid] * d_x_ptr[tid];
              ++d_Lij_yj;
            }
            d_y += number_of_grid_cells;
          }
        }
        // Backward Substitution
        {
          // d_y will be x_elem in the CPU implementation
          d_y = d_x + number_of_elements - number_of_grid_cells;
          for (auto i = 0; i < d_nUij_Uii_size; ++i)
          {
            const std::size_t j_lim = d_nUij_Uii[i].first;
            for (auto j = 0; j < j_lim; ++j)
            {
              auto d_ALU_ptr = d_ALU + (*d_Uij_xj).first;
              auto d_x_ptr = d_x + (*d_Uij_xj).second * number_of_grid_cells;
              d_y[tid] -= d_ALU_ptr[tid] * d_x_ptr[tid];
              ++d_Uij_xj;
            }
            auto d_ALU_ptr = d_ALU + d_nUij_Uii[i].second;
            d_y[tid] /= d_ALU_ptr[tid];
            d_y -= number_of_grid_cells;
          }
        }
      }
    }

    /// This is the function that will copy the constant data
    ///   members of class "CudaLinearSolver" to the device
    LinearSolverInPlaceParam CopyConstData(LinearSolverInPlaceParam& hoststruct)
    {
      /// Calculate the memory space of each constant data member
      size_t nLij_bytes = sizeof(size_t) * hoststruct.nLij_size_;
      size_t Lij_yj_bytes = sizeof(std::pair<size_t, size_t>) * hoststruct.Lij_yj_size_;
      size_t nUij_Uii_bytes = sizeof(std::pair<size_t, size_t>) * hoststruct.nUij_Uii_size_;
      size_t Uij_xj_bytes = sizeof(std::pair<size_t, size_t>) * hoststruct.Uij_xj_size_;

      /// Create a struct whose members contain the addresses in the device memory.
      LinearSolverInPlaceParam devstruct;
      CHECK_CUDA_ERROR(
          hipMallocAsync(&(devstruct.nLij_), nLij_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.Lij_yj_), Lij_yj_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.nUij_Uii_), nUij_Uii_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.Uij_xj_), Uij_xj_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");

      /// Copy the data from host to device
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.nLij_,
              hoststruct.nLij_,
              nLij_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.Lij_yj_,
              hoststruct.Lij_yj_,
              Lij_yj_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.nUij_Uii_,
              hoststruct.nUij_Uii_,
              nUij_Uii_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.Uij_xj_,
              hoststruct.Uij_xj_,
              Uij_xj_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");

      devstruct.nLij_size_ = hoststruct.nLij_size_;
      devstruct.Lij_yj_size_ = hoststruct.Lij_yj_size_;
      devstruct.nUij_Uii_size_ = hoststruct.nUij_Uii_size_;
      devstruct.Uij_xj_size_ = hoststruct.Uij_xj_size_;

      return devstruct;
    }

    /// This is the function that will delete the constant data
    ///   members of class "CudaLinearSolver" on the device
    void FreeConstData(LinearSolverInPlaceParam& devstruct)
    {
      if (devstruct.nLij_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.nLij_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
      if (devstruct.Lij_yj_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.Lij_yj_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
      if (devstruct.nUij_Uii_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.nUij_Uii_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
      if (devstruct.Uij_xj_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.Uij_xj_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
    }

    void
    SolveKernelDriver(CudaMatrixParam& x_param, const CudaMatrixParam& ALU_param, const LinearSolverInPlaceParam& devstruct)
    {
      size_t number_of_blocks = (x_param.number_of_grid_cells_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      SolveKernel<<<number_of_blocks, BLOCK_SIZE, 0, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)>>>(
          x_param, ALU_param, devstruct);
    }
  }  // namespace cuda
}  // namespace micm