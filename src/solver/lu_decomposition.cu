#include "hip/hip_runtime.h"
#include <iostream> 
#include <micm/util/cuda_param> 
const BLOCK_SIZE = 320; 

struct decomposeDevice{
    double* A; 
    double* L; 
    double* U; 
    bool* do_aik; 
    size_t* aik; 
    bool* do_aki;
    size_t* aki;  
    size_t* uii; 
}; 



namespace micm{
    namespace cuda{
        __global__ void Decompose_kernel(){

        }
    
        void DecomposeKernelDriver(
            CUDAMatrixParam& matrix, 
            CUDASolverParam& solver){
            
            //create device pointers and allocate device memory 
            double* d_A; 
            double* d_L; 
            double* d_U; 
            bool* d_do_aik; 
            size_t* d_aik; 
            bool* d_do_aki;
            size_t* d_aki;  
            size_t* d_uii; 
            decomposeDevice* device; 

            hipMalloc(&d_A, sizeof(double)* matrix.A_size); 
            hipMalloc(&d_L, sizeof(double)* matrix.L_size); 
            hipMalloc(&d_U, sizeof(double)* matrix.U_size); 
            hipMalloc(&d_do_aik, sizeof(bool)* solver.do_aik_size); 
            hipMalloc(&d_aik, sizeof(size_t)* solver.aik_size); 
            hipMalloc(&d_do_aki, sizeof(bool)* solver.do_aki_size); 
            hipMalloc(&d_aki, sizeof(size_t)* solver.aki_size); 
            hipMalloc(&d_uii, sizeof(size_t)* solver.uii_size); 
            hipMalloc(&device, sizeof(decomposeDevice)); 

            //transfer data from host to device 
            hipMemcpy(d_A, matrix.A, sizeof(double)* matrix.A_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_L, matrix.L, sizeof(double)* matrix.L_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_U, matrix.U, sizeof(double)* matrix.U_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_do_aik, solver.do_aik, sizeof(bool)* solver.do_aik_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_aik, solver.aik, sizeof(size_t)* solver.aik_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_do_aki, solver.do_aki, sizeof(bool)* solver.do_aki_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_uii, solver.uii, sizeof(size_t)* solver.uii_size, hipMemcpyHostToDevice); 
            hipMemcpy(&(device->A),&d_A, sizeof(double*), hipMemcpyHostToDevice);
            hipMemcpy(&(device->L),&d_L, sizeof(double*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->U),&d_U, sizeof(double*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->do_aik), &d_do_aik, sizeof(bool*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->aik), &d_aik, sizeof(size_t*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->do_aki),&d_do_aki,sizeof(bool*),hipMemcpyHostToDevice); 
            hipMemcpy(&(device->aki),&d_aki, sizeof(size_t*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->uii), &d_uii, sizeof(size_t*), hipMemcpyHostToDevice);
            
            
        

            }
        


    }
}