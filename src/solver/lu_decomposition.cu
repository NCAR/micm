#include "hip/hip_runtime.h"
#include <iostream> 
#include <vector>
#include <micm/util/cuda_param.hpp> 
const size_t BLOCK_SIZE = 320; 
struct decomposeDevice{
    double* A; 
    double* L; 
    double* U; 
    bool* do_aik; 
    size_t* aik; 
    bool* do_aki;
    size_t* aki;  
    size_t* uii; 
    std::pair<size_t,size_t>* niLU;
    std::pair<size_t, size_t>* uik_nkj; 
    std::pair<size_t, size_t>* lij_ujk;
    std::pair<size_t, size_t>* lki_nkj; 
    std::pair<size_t, size_t>* lkj_uji;
}; 
namespace micm{
    namespace cuda{
        __global__ void DecomposeKernel(
            decomposeDevice* device,
            size_t A_size)
        {
            size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
            double* A = device->A; 
            double* L = device->L;
            double* U = device->U;
            std::pair<size_t, size_t>* uik_nkj= device->uik_nkj;
            std::pair<size_t, size_t>* lij_ujk = device->lij_ujk;
            std::pair<size_t, size_t>* lki_nkj = device->lki_nkj;
            size_t do_aik_offset = 0; //boolean vector 
            size_t aik_offset = 0;
            size_t uik_nkj_offset = 0; 
            size_t lij_ujk_offset = 0; 
            size_t do_aki_offset = 0; //boolean vector 
            size_t aki_offset = 0; 
            size_t lki_nkj_offset = 0; 
            size_t lkj_uji_offset = 0; 
            size_t uii_offset = 0; 
            if (tid < A_size){
                for (auto& inLU : device.niLU){
                    //upper triangular matrix 
                    for (size_t iU = 0; iU < inLU.second; ++iU){
                        if(device->do_aik[++do_aik_offset]){
                            size_t U_idx = uik_nkj[uik_nkj_offset]->first + tid;
                            size_t A_idx =  device->aik[++aik_offset]+ tid; 
                            U[U_idx] = A[A_idx]; 
                        }
                        for (size_t ikj = 0; ikj < uik_nkj[uik_nkj_offset]->second; ++ikj){
                            
                            size_t L_idx = lij_ujk[lij_ujk_offset]->first + tid;
                            size_t U_idx_1 = uik_nkj[uik_nkj_offset]->first + tid; 
                            size_t U_idx_2 = lij_ujk[lij_ujk_offset]->second + tid; 
                            U[U_idx_1] -= L[L_idx] * U[U_idx_2]; 
                            ++lij_ujk_offset; 
                        }
                        ++uik_nkj_offset; 
                    }
                    //lower triangular matrix
                    L[lki_nkj[++lki_nkj_offset]->first + tid] = 1.0; 
                    for (size_t iL = 0; iL <inLU.first; ++iL){
                        if(device->do_aki[++do_aki_offset]){
                            size_t L_idx = lki_nkj[lkj_nkj_offset]->first + tid; 
                            size_t A_idx = aki->device[++aki_offset] + tid; 
                            L[L_idx] = A[A_idx]; 
                        }
                        //working in progress 
                        for(size_t ikj = 0; ikj < lki_nkj[lki_nkj_offset]->second;++ikj){
                            size_t L_idx_1 = lki_nkj[lki_nkj_offset]->first + tid;
                            size_t L_idx_2 = lkj_uji[lkj_uji_offset]->first + tid;
                            size_t U_idx = lkj_uji[lkj_uji_offset]->second + tid; 
                            ++lkj_uji_offset; 
                        }
                        size_t L_idx = lki_nkj[lki_nkj_offset]->first + tid; 
                        size_t U_idx = device->uii[uii_offset]+tid; 
                        L[L_idx]/=U[U_idx]; 
                        ++lki_nkj_offset; 
                        ++uii_offset; 
                    }
                }
            }
        }// end of kernel
    
        void DecomposeKernelDriver(
            CUDASparseMatrixParam& sparseMatrix, 
            CUDASolverParam& solver){
            //create device pointers and allocate device memory 
            double* d_A; 
            double* d_L; 
            double* d_U; 
            bool* d_do_aik; 
            size_t* d_aik; 
            bool* d_do_aki;
            size_t* d_aki;  
            size_t* d_uii; 
            std::pair<size_t, size_t>* d_niLU; 
            std::pair<size_t, size_t>* d_uik_nkj; 
            std::pair<size_t, size_t>* d_lij_ujk;
            std::pair<size_t, size_t>* d_lki_nkj; 
            std::pair<size_t, size_t>* d_lkj_uji;
            decomposeDevice* device; 
        
            hipMalloc(&d_A,sizeof(double)* sparseMatrix.A_size); 
            hipMalloc(&d_L,sizeof(double)* sparseMatrix.L_size); 
            hipMalloc(&d_U,sizeof(double)* sparseMatrix.U_size); 
            hipMalloc(&d_do_aik,sizeof(bool)* solver.do_aik_size); 
            hipMalloc(&d_aik,sizeof(size_t)* solver.aik_size); 
            hipMalloc(&d_do_aki,sizeof(bool)* solver.do_aki_size); 
            hipMalloc(&d_aki,sizeof(size_t)* solver.aki_size); 
            hipMalloc(&d_uii,sizeof(size_t)* solver.uii_size); 
            hipMalloc(&d_niLU,sizeof(std::pair<size_t, size_t>), solver.niLU_size); 
            hipMalloc(&d_uik_nkj,sizeof(std::pair<size_t, size_t>), solver.uik_nkj_size); 
            hipMalloc(&d_lij_ujk,sizeof(std::pair<size_t, size_t>), solver.lij_ujk_size); 
            hipMalloc(&d_lki_nkj,sizeof(std::pair<size_t, size_t>), solver.lki_nkj_size); 
            cudaMallco(&d_lkj_uji,sizeof(std::pair<size_t, size_t>), solver.lkj_uji_size);
            hipMalloc(&device, sizeof(decomposeDevice)); 

            //transfer data from host to device 
            hipMemcpy(d_A, sparseMatrix.A, sizeof(double)* sparseMatrix.A_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_L, sparseMatrix.L, sizeof(double)* sparseMatrix.L_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_U, sparseMatrix.U, sizeof(double)* sparseMatrix.U_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_do_aik, solver.do_aik, sizeof(char)* solver.do_aik_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_aik, solver.aik, sizeof(size_t)* solver.aik_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_do_aki, solver.do_aki, sizeof(char)* solver.do_aki_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_uii, solver.uii, sizeof(size_t)* solver.uii_size, hipMemcpyHostToDevice);       
            hipMemcpy(d_niLU, solver.niLU, sizeof(std::pair<size_t, size_t>)*solver.niLU_size, hipMemcpyHostToDevice); 
            hipMemcpy(d_uik_nkj, solver.uik_nkj, sizeof(std::pair<size_t, size_t>)*solver.uik_nkj_size, hipMemcpyHostToDevice);
            hipMemcpy(d_lij_ujk, solver.lij_ujk, sizeof(std::pair<size_t, size_t>)*solver.lij_ujk_size, hipMemcpyHostToDevice);
            hipMemcpy(d_lki_nkj, solver.lki_nkj, sizeof(std::pair<size_t, size_t>)*solver.lki_nkj_size, hipMemcpyHostToDevice);
            hipMemcpy(d_lkj_uji, solver.lkj_uji, sizeof(std::pair<size_t, size_t>)*solver.lkj_uji_size, hipMemcpyHostToDevice);

            hipMemcpy(&(device->A),&d_A, sizeof(double*), hipMemcpyHostToDevice);
            hipMemcpy(&(device->L),&d_L, sizeof(double*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->U),&d_U, sizeof(double*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->do_aik), &d_do_aik, sizeof(bool*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->aik), &d_aik, sizeof(size_t*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->do_aki),&d_do_aki,sizeof(bool*),hipMemcpyHostToDevice); 
            hipMemcpy(&(device->aki),&d_aki, sizeof(size_t*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->uii), &d_uii, sizeof(size_t*), hipMemcpyHostToDevice);
            hipMemcpy(&(device->niLU), &d_niLU, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice);
            hipMemcpy(&(device->uik_nkj), &d_uik_nkj, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->lij_ujk), &d_lij_ujk, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->lki_nkj), &d_lki_nkj, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice); 
            hipMemcpy(&(device->lkj_uji,), &d_lkj_uji, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice); 
            
            size_t num_block = (sparseMatrix.A_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
            size_t A_size = sparseMatrix.A_size; 
            //call kernel
            DecomposeKernel<<<BLOCK_SIZE, num_block>>>(device, A_size); 

        //clean up 
        hipFree(d_A); 
        hipFree(d_L); 
        hipFree(d_U); 
        hipFree(d_do_aik); 
        hipFree(d_aik);
        hipFree(d_do_aki); 
        hipFree(d_aki); 
        hipFree(d_uii); 
        hipFree(device); 
    }//end kernelDriver
 }//end cuda 
}//end micm