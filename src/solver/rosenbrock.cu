#include "hip/hip_runtime.h"
#pragma once
#include <chrono>
#include <vector>
#include <iostream>
#include <micm/util/cuda_param.hpp>

namespace micm{
    namespace cuda{
        __global__ void AlphaMinusJacobianKernel(size_t n_grids,
                                                double* d_jacobian,
                                                size_t* d_jacobian_diagonal_elements,
                                                size_t jacobian_diagonal_elements_size,
                                                double alpha)
{
        size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < n_grids)
    {
        for (int j = 0; j < jacobian_diagonal_elements_size; j++)
        {
            //printf("j: %d\n", j); 
            size_t jacobian_index = d_jacobian_diagonal_elements[j];
            printf("jacobian index: %d\n", d_jacobian_diagonal_elements[j]); 
            d_jacobian[jacobian_index + tid] += alpha; 
        }
    } 
}
        
        void AlphaMinusJacobianDriver(
                        CudaSparseMatrixParam& sparseMatrix,
                        
                        double alpha)
    {
        //device pointers
        std::cout<< "element size: "<<sparseMatrix.jacobian_diagonal_elements_size_<<std::endl; 
        
        
        double* d_jacobian;
        size_t* d_jacobian_diagonal_elements; 
        hipMalloc(&d_jacobian, sizeof(double)* sparseMatrix.jacobian_size_); 
        hipMalloc(&d_jacobian_diagonal_elements, sizeof(size_t)*sparseMatrix.jacobian_diagonal_elements_size_);
        hipMemcpy(d_jacobian, sparseMatrix.jacobian_, sparseMatrix.jacobian_size_, hipMemcpyHostToDevice); 
        hipMemcpy(d_jacobian_diagonal_elements, sparseMatrix.jacobian_diagonal_elements_, sparseMatrix.jacobian_diagonal_elements_size_, hipMemcpyHostToDevice);
        
        
        //kernel call
        size_t num_block = (sparseMatrix.n_grids_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
        AlphaMinusJacobianKernel<<<num_block, BLOCK_SIZE>>>(sparseMatrix.n_grids_,
                                d_jacobian,  
                                d_jacobian_diagonal_elements,
                                sparseMatrix.jacobian_diagonal_elements_size_,
                                alpha);
        
        hipDeviceSynchronize();
        hipMemcpy(sparseMatrix.jacobian_, d_jacobian, sparseMatrix.jacobian_size_, hipMemcpyDeviceToHost);
        hipFree(d_jacobian);
        hipFree(d_jacobian_diagonal_elements);
    }

    }// end cuda
}// end mimc 
