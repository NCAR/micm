#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0
#include <iostream>
#include <micm/solver/rosenbrock_solver_parameters.hpp>
#include <micm/util/cuda_param.hpp>
#include <micm/util/internal_error.hpp>

#include "hipblas.h"

namespace micm
{
  namespace cuda
  {
    /// CUDA kernel to compute alpha - J[i] for each element i at the diagnoal of Jacobian matrix
    __global__ void
    AlphaMinusJacobianKernel(CudaMatrixParam jacobian_param, const double alpha, const CudaRosenbrockSolverParam devstruct)
    {
      // Calculate global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;

      // Local device variables
      double* d_jacobian = jacobian_param.d_data_;
      size_t quotient, index_as_remainder;
      const size_t number_of_diagonal_elements = devstruct.jacobian_diagonal_elements_size_;
      const size_t number_of_grid_cells = jacobian_param.number_of_grid_cells_;

      if (tid < number_of_grid_cells * number_of_diagonal_elements)
      {
        quotient = tid / number_of_grid_cells;
        index_as_remainder = tid - number_of_grid_cells * quotient;  // % operator may be more expensive
        d_jacobian[devstruct.jacobian_diagonal_elements_[quotient] + index_as_remainder] += alpha;
      }
    }

    /// This is the function that will allocate device memory
    ///   and copy const data for data members of class "CudaRosenbrockSolverParam"
    CudaRosenbrockSolverParam CopyConstData(CudaRosenbrockSolverParam& hoststruct)
    {
      /// Calculate the memory space of each constant data member
      size_t jacobian_diagonal_elements_bytes = sizeof(size_t) * hoststruct.jacobian_diagonal_elements_size_;

      /// Calculate the memory space of each temporary variable
      size_t errors_bytes = sizeof(double) * hoststruct.errors_size_;
      size_t tolerance_bytes = sizeof(double) * hoststruct.absolute_tolerance_size_;

      /// Create a struct whose members contain the addresses in the device memory.
      CudaRosenbrockSolverParam devstruct;
      hipMalloc(&(devstruct.errors_input_), errors_bytes);
      hipMalloc(&(devstruct.errors_output_), errors_bytes);
      hipMalloc(&(devstruct.jacobian_diagonal_elements_), jacobian_diagonal_elements_bytes);
      hipMalloc(&(devstruct.absolute_tolerance_), tolerance_bytes);

      /// Copy the data from host to device
      hipMemcpy(
          devstruct.jacobian_diagonal_elements_,
          hoststruct.jacobian_diagonal_elements_,
          jacobian_diagonal_elements_bytes,
          hipMemcpyHostToDevice);

      hipMemcpy(devstruct.absolute_tolerance_, hoststruct.absolute_tolerance_, tolerance_bytes, hipMemcpyHostToDevice);

      devstruct.errors_size_ = hoststruct.errors_size_;
      devstruct.jacobian_diagonal_elements_size_ = hoststruct.jacobian_diagonal_elements_size_;
      devstruct.absolute_tolerance_size_ = hoststruct.absolute_tolerance_size_;

      return devstruct;
    }

    /// This is the function that will delete the constant data
    ///   members and temporary variables of class "CudaLuDecomposition" on the device
    void FreeConstData(CudaRosenbrockSolverParam& devstruct)
    {
      hipFree(devstruct.errors_input_);
      hipFree(devstruct.errors_output_);
      hipFree(devstruct.jacobian_diagonal_elements_);
      hipFree(devstruct.absolute_tolerance_);
    }

    // Specific CUDA device function to do reduction within a warp
    // Use volatile to prevent compiler optimization (caching in registers)
    // No need to synchronize threads in the same warp
    __device__ void warpReduce(volatile double* sdata, size_t tid)
    {
      if (BLOCK_SIZE >= 64)
        sdata[tid] += sdata[tid + 32];
      sdata[tid] += sdata[tid + 16];
      sdata[tid] += sdata[tid + 8];
      sdata[tid] += sdata[tid + 4];
      sdata[tid] += sdata[tid + 2];
      sdata[tid] += sdata[tid + 1];
    }

    // CUDA kernel to compute the scaled norm of the vector errors; CUDA kernel does not take reference as argument
    // Modified version from NVIDIA's reduction example:
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    __global__ void NormalizedErrorKernel(
        const CudaMatrixParam y_old_param,
        const CudaMatrixParam y_new_param,
        const RosenbrockSolverParameters ros_param,
        CudaRosenbrockSolverParam devstruct,
        const size_t n,
        bool is_first_call)
    {
      double* d_y_old = y_old_param.d_data_;
      double* d_y_new = y_new_param.d_data_;
      double* d_errors_input = devstruct.errors_input_;
      double* d_errors_output = devstruct.errors_output_;
      const double* atol = devstruct.absolute_tolerance_;
      const double rtol = ros_param.relative_tolerance_;
      const size_t number_of_grid_cells = y_old_param.number_of_grid_cells_;

      // Declares a dynamically-sized shared memory array.
      // The size of this array is determined at runtime when the kernel is launched.
      // Shared memory is shared among all threads within the same block.
      extern __shared__ double sdata[];

      // Calculate local thread ID within a threadblock
      size_t l_tid = threadIdx.x;

      // Calculate global thread ID
      size_t g_tid = blockIdx.x * (BLOCK_SIZE * 2) + threadIdx.x;

      if (is_first_call)
      {
        // Local device variables
        double d_ymax, d_scale;

        // Load two elements by one thread and do first add of reduction
        sdata[l_tid] = 0.0;
        for (int i = 0; i < 2; ++i)
        {
          if (g_tid < n)
          {
            d_ymax = max(fabs(d_y_old[g_tid]), fabs(d_y_new[g_tid]));
            d_scale = atol[g_tid / number_of_grid_cells] + rtol * d_ymax;
            d_errors_input[g_tid] = d_errors_input[g_tid] * d_errors_input[g_tid] / (d_scale * d_scale);
            sdata[l_tid] += d_errors_input[g_tid];
          }
          g_tid += BLOCK_SIZE;
        }
        __syncthreads();
      }
      else
      {
        // Load two elements by one thread and do first add of reduction
        // Access the d_errors array directly if it is not the first call
        sdata[l_tid] = 0.0;
        if (g_tid < n)
          sdata[l_tid] += d_errors_input[g_tid];
        g_tid += BLOCK_SIZE;
        if (g_tid < n)
          sdata[l_tid] += d_errors_input[g_tid];
        __syncthreads();
      }

      // Start at 1/2 block stride, do the add, and divide by two each iteration
      if (BLOCK_SIZE >= 1024)
      {
        if (l_tid < 512)
        {
          sdata[l_tid] += sdata[l_tid + 512];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 512)
      {
        if (l_tid < 256)
        {
          sdata[l_tid] += sdata[l_tid + 256];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 256)
      {
        if (l_tid < 128)
        {
          sdata[l_tid] += sdata[l_tid + 128];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 128)
      {
        if (l_tid < 64)
        {
          sdata[l_tid] += sdata[l_tid + 64];
        }
        __syncthreads();
      }
      if (l_tid < 32)
        warpReduce(sdata, l_tid);

      // Let the thread 0 of this threadblock write its result to output array, inexed by this threadblock
      if (l_tid == 0)
        d_errors_output[blockIdx.x] = sdata[0];
    }

    // CUDA kernel to compute the scaled vectors; prepare the input for cublas call later
    __global__ void ScaledErrorKernel(
        const CudaMatrixParam y_old_param,
        const CudaMatrixParam y_new_param,
        const RosenbrockSolverParameters ros_param,
        CudaRosenbrockSolverParam devstruct)
    {
      // Local device variables
      double d_ymax, d_scale;
      double* d_y_old = y_old_param.d_data_;
      double* d_y_new = y_new_param.d_data_;
      double* d_errors = devstruct.errors_input_;
      const double* atol = devstruct.absolute_tolerance_;
      double rtol = ros_param.relative_tolerance_;
      const size_t num_elements = devstruct.errors_size_;
      const size_t number_of_grid_cells = y_old_param.number_of_grid_cells_;

      // Calculate global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;
      if (tid < num_elements)
      {
        d_ymax = max(fabs(d_y_old[tid]), fabs(d_y_new[tid]));
        d_scale = atol[tid / number_of_grid_cells] + rtol * d_ymax;
        d_errors[tid] = d_errors[tid] / d_scale;
      }
    }

    // Host code that will launch the AlphaMinusJacobian CUDA kernel
    void AlphaMinusJacobianDriver(
        CudaMatrixParam& jacobian_param,
        const double& alpha,
        const CudaRosenbrockSolverParam& devstruct)
    {
      size_t number_of_blocks =
          (devstruct.jacobian_diagonal_elements_size_ * jacobian_param.number_of_grid_cells_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      AlphaMinusJacobianKernel<<<number_of_blocks, BLOCK_SIZE>>>(jacobian_param, alpha, devstruct);
      hipDeviceSynchronize();
    }

    // Host code that will launch the NormalizedError CUDA kernel
    double NormalizedErrorDriver(
        const CudaMatrixParam& y_old_param,
        const CudaMatrixParam& y_new_param,
        const CudaMatrixParam& errors_param,
        const RosenbrockSolverParameters& ros_param,
        hipblasHandle_t handle,
        CudaRosenbrockSolverParam devstruct)
    {
      double normalized_error;
      const size_t number_of_elements = devstruct.errors_size_;

      if (number_of_elements != errors_param.number_of_elements_)
      {
        std::string msg = "mismatch in normalized error arrays. Expected: " + std::to_string(number_of_elements) +
                          " but got: " + std::to_string(errors_param.number_of_elements_);
        INTERNAL_ERROR(msg.c_str());
      }
      hipError_t err = hipMemcpy(
          devstruct.errors_input_, errors_param.d_data_, sizeof(double) * number_of_elements, hipMemcpyDeviceToDevice);

      if (number_of_elements > 1000000)
      {
        // call cublas APIs
        size_t number_of_blocks = (number_of_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
        ScaledErrorKernel<<<number_of_blocks, BLOCK_SIZE>>>(y_old_param, y_new_param, ros_param, devstruct);
        // call cublas function to perform the norm:
        // https://docs.nvidia.com/cuda/cublas/index.html?highlight=dnrm2#cublas-t-nrm2
        hipblasStatus_t stat = hipblasDnrm2(handle, number_of_elements, devstruct.errors_input_, 1, &normalized_error);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
          ThrowInternalError(MicmInternalErrc::Cublas, __FILE__, __LINE__, cublasGetStatusString(stat));
        }
        normalized_error = normalized_error * std::sqrt(1.0 / number_of_elements);
      }
      else
      {
        // call CUDA implementation
        size_t number_of_blocks = std::ceil(std::ceil(number_of_elements * 1.0 / BLOCK_SIZE) / 2.0);
        number_of_blocks = number_of_blocks < 1 ? 1 : number_of_blocks;
        size_t new_number_of_blocks;
        bool is_first_call = true;

        // Kernel call
        NormalizedErrorKernel<<<number_of_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
            y_old_param, y_new_param, ros_param, devstruct, number_of_elements, is_first_call);
        is_first_call = false;
        while (number_of_blocks > 1)
        {
          std::swap(devstruct.errors_input_, devstruct.errors_output_);
          // Update grid size
          new_number_of_blocks = std::ceil(std::ceil(number_of_blocks * 1.0 / BLOCK_SIZE) / 2.0);
          if (new_number_of_blocks <= 1)
          {
            NormalizedErrorKernel<<<1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
                y_old_param, y_new_param, ros_param, devstruct, number_of_blocks, is_first_call);
            break;
          }
          NormalizedErrorKernel<<<new_number_of_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
              y_old_param, y_new_param, ros_param, devstruct, number_of_blocks, is_first_call);
          number_of_blocks = new_number_of_blocks;
        }
        hipDeviceSynchronize();

        hipMemcpy(&normalized_error, &devstruct.errors_output_[0], sizeof(double), hipMemcpyDeviceToHost);
        normalized_error = std::sqrt(normalized_error / number_of_elements);
      }  // end of if-else for CUDA/CUBLAS implementation
      return std::max(normalized_error, 1.0e-10);
    }  // end of NormalizedErrorDriver function
  }    // namespace cuda
}  // namespace micm
