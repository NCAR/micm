#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#pragma once
#include <chrono>
#include <iostream>
#include <micm/util/cuda_param.hpp>
#include <vector>

namespace micm
{
  namespace cuda
  {
    __global__ void AlphaMinusJacobianKernel(
        size_t n_grids,
        double* d_jacobian,
        size_t* d_jacobian_diagonal_elements,
        size_t jacobian_diagonal_elements_size,
        double alpha)
    {
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      if (tid < n_grids)
      {
        for (int j = 0; j < jacobian_diagonal_elements_size; j++)
        {
          size_t jacobian_index = d_jacobian_diagonal_elements[j];
          d_jacobian[jacobian_index + tid] += alpha;
        }
      }
    }

    std::chrono::nanoseconds AlphaMinusJacobianDriver(
        CudaSparseMatrixParam& sparseMatrix,
        const std::vector<size_t> jacobian_diagonal_elements,
        double alpha)
    {
      // device pointers
      double* d_jacobian;
      size_t* d_jacobian_diagonal_elements;
      hipMalloc(&d_jacobian, sizeof(double) * sparseMatrix.jacobian_size_);
      hipMalloc(&d_jacobian_diagonal_elements, sizeof(size_t) * jacobian_diagonal_elements.size());
      hipMemcpy(d_jacobian, sparseMatrix.jacobian_, sizeof(double) * sparseMatrix.jacobian_size_, hipMemcpyHostToDevice);
      hipMemcpy(
          d_jacobian_diagonal_elements,
          jacobian_diagonal_elements.data(),
          sizeof(size_t) * jacobian_diagonal_elements.size(),
          hipMemcpyHostToDevice);

      // kernel call
      size_t num_block = (sparseMatrix.n_grids_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      auto startTime = std::chrono::high_resolution_clock::now();
      AlphaMinusJacobianKernel<<<num_block, BLOCK_SIZE>>>(
          sparseMatrix.n_grids_, d_jacobian, d_jacobian_diagonal_elements, jacobian_diagonal_elements.size(), alpha);

      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);
      hipMemcpy(sparseMatrix.jacobian_, d_jacobian, sizeof(double) * sparseMatrix.jacobian_size_, hipMemcpyDeviceToHost);
      hipFree(d_jacobian);
      hipFree(d_jacobian_diagonal_elements);

      return kernel_duration;
    }

  }  // namespace cuda
}  // namespace micm
