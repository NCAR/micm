#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <chrono>
#include <iostream>
#include <micm/solver/rosenbrock_solver_parameters.hpp>
#include <micm/util/cuda_param.hpp>
#include <vector>

#include "hipblas.h"

namespace micm
{
  namespace cuda
  {
    /// CUDA kernel to compute alpha - J[i] for each element i at the diagnoal of matrix J
    __global__ void AlphaMinusJacobianKernel(double* d_jacobian, const double alpha, CudaRosenbrockSolverParam devstruct)
    {
      // Global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;

      // Local variables
      size_t quotient, index_as_remainder;
      const size_t num_diagonal_elements = devstruct.jacobian_diagonal_elements_size_;
      const size_t num_grid_cells = devstruct.num_grid_cells_;

      if (tid < num_grid_cells * num_diagonal_elements)
      {
        quotient = tid / num_diagonal_elements;
        index_as_remainder = tid - num_diagonal_elements * quotient;  // % operator may be more expensive
        d_jacobian[devstruct.jacobian_diagonal_elements_[index_as_remainder] + quotient] += alpha;
      }
    }

    /// This is the function that will allocate device memory
    ///   and copy const data for data members of class "CudaRosenbrockSolverParam"
    CudaRosenbrockSolverParam CopyConstData(CudaRosenbrockSolverParam& hoststruct)
    {
      /// Calculate the memory space of each constant data member
      size_t jacobian_diagonal_elements_bytes = sizeof(size_t) * hoststruct.jacobian_diagonal_elements_size_;

      /// Calculate the memory space of each temporary variable
      size_t errors_bytes = sizeof(double) * hoststruct.errors_size_;

      /// Create a struct whose members contain the addresses in the device memory.
      CudaRosenbrockSolverParam devstruct;
      hipMalloc(&(devstruct.errors_input_), errors_bytes);
      hipMalloc(&(devstruct.errors_output_), errors_bytes);
      hipMalloc(&(devstruct.jacobian_diagonal_elements_), jacobian_diagonal_elements_bytes);

      /// Copy the data from host to device
      hipMemcpy(
          devstruct.jacobian_diagonal_elements_,
          hoststruct.jacobian_diagonal_elements_,
          jacobian_diagonal_elements_bytes,
          hipMemcpyHostToDevice);

      devstruct.num_grid_cells_ = hoststruct.num_grid_cells_;
      devstruct.errors_size_ = hoststruct.errors_size_;
      devstruct.jacobian_diagonal_elements_size_ = hoststruct.jacobian_diagonal_elements_size_;

      return devstruct;
    }

    /// This is the function that will delete the constant data
    ///   members and temporary variables of class "CudaLuDecomposition" on the device
    void FreeConstData(CudaRosenbrockSolverParam& devstruct)
    {
      hipFree(devstruct.errors_input_);
      hipFree(devstruct.errors_output_);
      hipFree(devstruct.jacobian_diagonal_elements_);
    }

    // Specific CUDA device function to do reduction within a warp
    // Use volatile to prevent compiler optimization (caching in registers)
    // No need to synchronize threads in the same warp
    __device__ void warpReduce(volatile double* sdata, size_t tid)
    {
      if (BLOCK_SIZE >= 64)
        sdata[tid] += sdata[tid + 32];
      sdata[tid] += sdata[tid + 16];
      sdata[tid] += sdata[tid + 8];
      sdata[tid] += sdata[tid + 4];
      sdata[tid] += sdata[tid + 2];
      sdata[tid] += sdata[tid + 1];
    }

    // CUDA kernel to compute the scaled norm of the vector errors; CUDA kernel does not take reference as argument
    // Modified version from NVIDIA's reduction example:
    // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
    __global__ void NormalizedErrorKernel(
        const CudaMatrixParam y_old_param,
        const CudaMatrixParam y_new_param,
        const RosenbrockSolverParameters ros_param,
        CudaRosenbrockSolverParam devstruct,
        const size_t n,
        bool is_first_call)
    {
      double* d_y_old = y_old_param.d_data_;
      double* d_y_new = y_new_param.d_data_;
      double* d_errors_input = devstruct.errors_input_;
      double* d_errors_output = devstruct.errors_output_;
      const double* atol = devstruct.absolute_tolerance_;
      const double rtol = ros_param.relative_tolerance_;

      // Declares a dynamically-sized shared memory array.
      // The size of this array is determined at runtime when the kernel is launched.
      // Shared memory is shared among all threads within the same block.
      extern __shared__ double sdata[];

      // Local thread ID within a threadblock
      size_t l_tid = threadIdx.x;

      // Global thread ID
      size_t g_tid = blockIdx.x * (BLOCK_SIZE * 2) + threadIdx.x;

      if (is_first_call)
      {
        // Temporary device variables
        double d_ymax, d_scale;

        // Load two elements by one thread and do first add of reduction
        sdata[l_tid] = 0.0;
        for (int i = 0; i < 2; ++i)
        {
          if (g_tid < n)
          {
            d_ymax = max(fabs(d_y_old[g_tid]), fabs(d_y_new[g_tid]));
            d_scale = atol[g_tid] + rtol * d_ymax;
            d_errors_input[g_tid] = d_errors_input[g_tid] * d_errors_input[g_tid] / (d_scale * d_scale);
            sdata[l_tid] += d_errors_input[g_tid];
          }
          g_tid += BLOCK_SIZE;
        }
        __syncthreads();
      }
      else
      {
        // Load two elements by one thread and do first add of reduction
        // Access the d_errors array directly if it is not the first call
        sdata[l_tid] = 0.0;
        if (g_tid < n)
          sdata[l_tid] += d_errors_input[g_tid];
        g_tid += BLOCK_SIZE;
        if (g_tid < n)
          sdata[l_tid] += d_errors_input[g_tid];
        __syncthreads();
      }

      // Start at 1/2 block stride, do the add, and divide by two each iteration
      if (BLOCK_SIZE >= 1024)
      {
        if (l_tid < 512)
        {
          sdata[l_tid] += sdata[l_tid + 512];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 512)
      {
        if (l_tid < 256)
        {
          sdata[l_tid] += sdata[l_tid + 256];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 256)
      {
        if (l_tid < 128)
        {
          sdata[l_tid] += sdata[l_tid + 128];
        }
        __syncthreads();
      }
      if (BLOCK_SIZE >= 128)
      {
        if (l_tid < 64)
        {
          sdata[l_tid] += sdata[l_tid + 64];
        }
        __syncthreads();
      }
      if (l_tid < 32)
        warpReduce(sdata, l_tid);

      // Let the thread 0 of this threadblock write its result to output array, inexed by this threadblock
      if (l_tid == 0)
        d_errors_output[blockIdx.x] = sdata[0];
    }

    // CUDA kernel to compute the scaled vectors; prepare the input for cublas call later
    __global__ void ScaledErrorKernel(
        const CudaMatrixParam y_old_param,
        const CudaMatrixParam y_new_param,
        const RosenbrockSolverParameters ros_param,
        CudaRosenbrockSolverParam devstruct)
    {
      // Temporary device variables
      double d_ymax, d_scale;
      double* d_y_old = y_old_param.d_data_;
      double* d_y_new = y_new_param.d_data_;
      double* d_errors = devstruct.errors_input_;
      double* atol = devstruct.absolute_tolerance_;
      double rtol = ros_param.relative_tolerance_;
      const size_t num_elements = devstruct.errors_size_;

      // Global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;
      if (tid < num_elements)
      {
        d_ymax = max(fabs(d_y_old[tid]), fabs(d_y_new[tid]));
        d_scale = atol[tid] + rtol * d_ymax;
        d_errors[tid] = d_errors[tid] / d_scale;
      }
    }

    // Host code that will launch the AlphaMinusJacobian CUDA kernel
    void AlphaMinusJacobianDriver(
        double* h_jacobian,
        const size_t num_elements,
        const double alpha,
        const CudaRosenbrockSolverParam& devstruct)
    {
      // device pointers (will not be needed after adding the CudaSparseMatrix class)
      double* d_jacobian;
      hipMalloc(&d_jacobian, sizeof(double) * num_elements);
      hipMemcpy(d_jacobian, h_jacobian, sizeof(double) * num_elements, hipMemcpyHostToDevice);

      // kernel call
      size_t num_blocks =
          (devstruct.jacobian_diagonal_elements_size_ * devstruct.num_grid_cells_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      AlphaMinusJacobianKernel<<<num_blocks, BLOCK_SIZE>>>(d_jacobian, alpha, devstruct);

      hipDeviceSynchronize();
      hipMemcpy(h_jacobian, d_jacobian, sizeof(double) * num_elements, hipMemcpyDeviceToHost);
      hipFree(d_jacobian);
    }

    // Host code that will launch the NormalizedError CUDA kernel
    double NormalizedErrorDriver(
        const CudaMatrixParam& y_old_param,
        const CudaMatrixParam& y_new_param,
        const CudaMatrixParam& errors_param,
        const RosenbrockSolverParameters& ros_param,
        hipblasHandle_t handle,
        CudaRosenbrockSolverParam devstruct)
    {
      double normalized_error;
      const size_t num_elements = devstruct.errors_size_;

      if (devstruct.errors_size_ != errors_param.number_of_elements_)
      {
        throw std::runtime_error("devstruct.errors_input_ and errors_param have different sizes.");
      }
      hipError_t err =
          hipMemcpy(devstruct.errors_input_, errors_param.d_data_, sizeof(double) * num_elements, hipMemcpyDeviceToDevice);

      if (num_elements > 1000000)
      {
        // call cublas APIs
        size_t num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
        ScaledErrorKernel<<<num_blocks, BLOCK_SIZE>>>(y_old_param, y_new_param, ros_param, devstruct);
        // call cublas function to perform the norm:
        // https://docs.nvidia.com/cuda/cublas/index.html?highlight=dnrm2#cublas-t-nrm2
        hipblasStatus_t stat = hipblasDnrm2(handle, num_elements, devstruct.errors_input_, 1, &normalized_error);
        if (stat != HIPBLAS_STATUS_SUCCESS)
        {
          printf(cublasGetStatusString(stat));
          throw std::runtime_error("Error while calling hipblasDnrm2.");
        }
        normalized_error = normalized_error * std::sqrt(1.0 / num_elements);
      }
      else
      {
        // call CUDA implementation
        size_t num_blocks = std::ceil(std::ceil(num_elements * 1.0 / BLOCK_SIZE) / 2.0);
        num_blocks = num_blocks < 1 ? 1 : num_blocks;
        size_t new_blocks;
        bool is_first_call;

        is_first_call = true;
        // Kernel call
        NormalizedErrorKernel<<<num_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
            y_old_param, y_new_param, ros_param, devstruct, num_elements, is_first_call);
        is_first_call = false;
        while (num_blocks > 1)
        {
          std::swap(devstruct.errors_input_, devstruct.errors_output_);
          // Update grid size
          new_blocks = std::ceil(std::ceil(num_blocks * 1.0 / BLOCK_SIZE) / 2.0);
          if (new_blocks <= 1)
          {
            NormalizedErrorKernel<<<1, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
                y_old_param, y_new_param, ros_param, devstruct, num_blocks, is_first_call);
            break;
          }
          NormalizedErrorKernel<<<new_blocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(double)>>>(
              y_old_param, y_new_param, ros_param, devstruct, num_blocks, is_first_call);
          num_blocks = new_blocks;
        }
        hipDeviceSynchronize();

        hipMemcpy(&normalized_error, &devstruct.errors_output_[0], sizeof(double), hipMemcpyDeviceToHost);
        normalized_error = std::sqrt(normalized_error / num_elements);
      }  // end of if-else for CUDA/CUBLAS implementation
      return std::max(normalized_error, 1.0e-10);
    }  // end of NormalizedErrorDriver function
  }    // namespace cuda
}  // namespace micm
