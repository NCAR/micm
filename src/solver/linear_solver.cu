#include "hip/hip_runtime.h"
// Copyright (C) 2023 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0

#include <iostream> 
#include <vector>
#include <chrono>
#include <micm/util/cuda_param.hpp> 
struct SolveDevice{
    std::pair<size_t, size_t>* nLij_Lii_;
    std::pair<size_t, size_t>* Lij_yj_; 
    std::pair<size_t, size_t>* nUij_Uii_;
    std::pair<size_t, size_t>* Uij_xj_;
    double* lower_matrix_;
    double* upper_matrix_; 
    double* b_; 
    double* x_;
};
namespace micm{
    namespace cuda{
__global__ void SolveKernel(SolveDevice* device, 
                            size_t n_grids,
                            size_t b_column_counts,
                            size_t x_column_counts,
                            size_t nLij_Lii_size,
                            size_t nUij_Uii_size)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    double* b = device->b_;
    double* x = device->x_;
    double* y = device->x_; //Alias x for consistency with equation, but to reuse memory
    double* lower_matrix = device->lower_matrix_;
    double* upper_matrix = device->upper_matrix_;
    std::pair<size_t, size_t>* nLij_Lii = device->nLij_Lii_;
    std::pair<size_t, size_t>* Lij_yj = device->Lij_yj_;
    std::pair<size_t, size_t>* nUij_Uii = device->nUij_Uii_;
    std::pair<size_t, size_t>* Uij_xj = device->Uij_xj_;

   if (tid < n_grids)
   { 
        size_t b_column_index = 0;
        size_t x_column_index = 0;
        size_t y_column_index = 0;
        size_t b_column_backward_index = b_column_counts -1;
        size_t x_column_backward_index = x_column_counts -1;
        size_t Lij_yj_index = 0; 
        size_t Uij_xj_index = 0;
       
        for (size_t j = 0; j < nLij_Lii_size; ++j)
        {
            auto& nLij_Lii_element = nLij_Lii[j]; 
            y[y_column_index * n_grids + tid] = b[b_column_index++ * n_grids + tid]; 
            for (size_t i = 0; i < nLij_Lii_element.first; ++i)
            {
                size_t lower_matrix_index = Lij_yj[Lij_yj_index].first + tid;
                size_t y_index = Lij_yj[Lij_yj_index].second * n_grids + tid; 
                y[y_column_index * n_grids + tid] -= lower_matrix[lower_matrix_index] * y[y_index];
                ++Lij_yj_index;  
            }
            y[y_column_index++ * n_grids + tid] /= lower_matrix[nLij_Lii_element.second + tid]; 
        }
        
        for (size_t k = 0; k < nUij_Uii_size; ++k)
        {   
            auto& nUij_Uii_element = nUij_Uii[k]; 
        
            for (size_t i = 0; i < nUij_Uii_element.first; ++i)
            {
                size_t upper_matrix_index = Uij_xj[Uij_xj_index].first + tid;
                size_t x_index = Uij_xj[Uij_xj_index].second * n_grids + tid;
                x[x_column_backward_index * n_grids + tid] -= upper_matrix[upper_matrix_index] * x[x_index];
                ++Uij_xj_index;
            }
            x[x_column_backward_index * n_grids + tid] /= upper_matrix[nUij_Uii_element.second + tid];
            
            if (x_column_backward_index != 0)
            {
                --x_column_backward_index;
            }
        }
    }
}
    std::chrono::nanoseconds SolveKernelDriver(CudaLinearSolverParam& linearSolver,CudaSparseMatrixParam& sparseMatrix, CudaMatrixParam& denseMatrix)
    {
    //create device pointer
    std::pair<size_t, size_t>* d_nLij_Lii; 
    std::pair<size_t, size_t>* d_Lij_yj; 
    std::pair<size_t, size_t>* d_nUij_Uii; 
    std::pair<size_t, size_t>* d_Uij_xj;
    double* d_lower_matrix; 
    double* d_upper_matrix;
    double* d_b; 
    double* d_x;
    SolveDevice* device;

    //allocate device memory 
    hipMalloc(&d_nLij_Lii, sizeof(std::pair<size_t, size_t>)* linearSolver.nLij_Lii_size_); 
    hipMalloc(&d_Lij_yj, sizeof(std::pair<size_t, size_t>)* linearSolver.Lij_yj_size_); 
    hipMalloc(&d_nUij_Uii, sizeof(std::pair<size_t, size_t>)* linearSolver.nUij_Uii_size_);
    hipMalloc(&d_Uij_xj, sizeof(std::pair<size_t, size_t>)* linearSolver.Uij_xj_size_); 
    
    hipMalloc(&d_lower_matrix, sizeof(double)* sparseMatrix.lower_matrix_size_); 
    hipMalloc(&d_upper_matrix, sizeof(double)* sparseMatrix.upper_matrix_size_);
    hipMalloc(&d_b, sizeof(double)* denseMatrix.b_size_);
    hipMalloc(&d_x, sizeof(double)* denseMatrix.x_size_); 
    hipMalloc(&device, sizeof(SolveDevice));

    //transfer memory from host to device
    hipMemcpy(d_nLij_Lii, linearSolver.nLij_Lii_, sizeof(std::pair<size_t, size_t>)* linearSolver.nLij_Lii_size_,hipMemcpyHostToDevice);
    hipMemcpy(d_Lij_yj, linearSolver.Lij_yj_, sizeof(std::pair<size_t, size_t>)* linearSolver.Lij_yj_size_,hipMemcpyHostToDevice);
    hipMemcpy(d_nUij_Uii, linearSolver.nUij_Uii_, sizeof(std::pair<size_t, size_t>)* linearSolver.nUij_Uii_size_,hipMemcpyHostToDevice);
    hipMemcpy(d_Uij_xj, linearSolver.Uij_xj_, sizeof(std::pair<size_t, size_t>)* linearSolver.Uij_xj_size_, hipMemcpyHostToDevice);
    
    hipMemcpy(d_lower_matrix, sparseMatrix.lower_matrix_, sizeof(double)*sparseMatrix.lower_matrix_size_, hipMemcpyHostToDevice);
    hipMemcpy(d_upper_matrix, sparseMatrix.upper_matrix_, sizeof(double)*sparseMatrix.upper_matrix_size_, hipMemcpyHostToDevice);
    hipMemcpy(d_b, denseMatrix.b_, sizeof(double)* denseMatrix.b_size_, hipMemcpyHostToDevice);
    hipMemcpy(d_x, denseMatrix.x_, sizeof(double)* denseMatrix.x_size_, hipMemcpyHostToDevice);
    
    hipMemcpy(&(device->nLij_Lii_), &d_nLij_Lii, sizeof(std::pair<size_t, size_t>*),hipMemcpyHostToDevice);
    hipMemcpy(&(device->Lij_yj_), &d_Lij_yj, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice);
    hipMemcpy(&(device->nUij_Uii_), &d_nUij_Uii, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice);
    hipMemcpy(&(device->Uij_xj_), &d_Uij_xj, sizeof(std::pair<size_t, size_t>*), hipMemcpyHostToDevice);
    
    hipMemcpy(&(device->lower_matrix_), &d_lower_matrix, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(device->upper_matrix_), &d_upper_matrix, sizeof(double*), hipMemcpyHostToDevice);
    hipMemcpy(&(device->b_), &d_b, sizeof(double*), hipMemcpyHostToDevice); 
    hipMemcpy(&(device->x_),&d_x, sizeof(double*), hipMemcpyHostToDevice);
    
    //kernel call 
    size_t num_block = (denseMatrix.n_grids_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
    auto startTime = std::chrono::high_resolution_clock::now();
    SolveKernel<<<num_block, BLOCK_SIZE>>>(device, 
                                          denseMatrix.n_grids_, 
                                          denseMatrix.b_column_counts_, 
                                          denseMatrix.x_column_counts_,
                                          linearSolver.nLij_Lii_size_,
                                          linearSolver.nUij_Uii_size_);
    hipDeviceSynchronize();
    auto endTime = std::chrono::high_resolution_clock::now();
    auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);
    hipMemcpy(denseMatrix.x_, d_x, sizeof(double)* denseMatrix.x_size_, hipMemcpyDeviceToHost);

    //clean up 
    hipFree(d_nLij_Lii);
    hipFree(d_Lij_yj); 
    hipFree(d_nUij_Uii); 
    hipFree(d_Uij_xj);
    hipFree(d_lower_matrix); 
    hipFree(d_upper_matrix);
    hipFree(d_b); 
    hipFree(d_x);
    hipFree(device);
    return kernel_duration;
    }
  }//end cuda 
}// end micm 