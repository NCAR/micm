#include "hip/hip_runtime.h"
// Copyright (C) 2023-2025 University Corporation for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <micm/cuda/util/cuda_param.hpp>
#include <micm/cuda/util/cuda_util.cuh>

namespace micm
{
  namespace cuda
  {
    /// This is the CUDA kernel that performs LU decomposition on the device
    __global__ void DecomposeKernel(CudaMatrixParam ALU_param, const LuDecomposeParam devstruct)
    {
      // Calculate global thread ID
      size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;

      // Local device variables
      const std::tuple<std::size_t, std::size_t, std::size_t>* const __restrict__ d_aii_nji_nki = devstruct.aii_nji_nki_;
      const std::size_t* __restrict__ d_aji = devstruct.aji_;
      const std::pair<std::size_t, std::size_t>* __restrict__ d_aik_njk = devstruct.aik_njk_;
      const std::pair<std::size_t, std::size_t>* __restrict__ d_ajk_aji = devstruct.ajk_aji_;
      const std::size_t d_aii_nji_nki_size = devstruct.aii_nji_nki_size_;

      double* __restrict__ d_ALU = ALU_param.d_data_;
      const size_t number_of_grid_cells = ALU_param.number_of_grid_cells_;
      const size_t cuda_matrix_vector_length = ALU_param.vector_length_;
      const std::size_t local_tid = tid % cuda_matrix_vector_length;
      const std::size_t group_id = std::floor(static_cast<double>(tid) / cuda_matrix_vector_length);

      // Shift the index for different groups
      d_ALU += group_id * devstruct.number_of_non_zeros_ * cuda_matrix_vector_length;

      if (tid < number_of_grid_cells)
      {
        for (std::size_t i = 0; i < d_aii_nji_nki_size; ++i)
        {
          auto& d_aii_nji_nki_elem = d_aii_nji_nki[i];
          auto d_Aii = d_ALU + std::get<0>(d_aii_nji_nki_elem);
          auto d_Aii_inverse = 1.0 / d_Aii[local_tid];
          for (std::size_t ij = 0; ij < std::get<1>(d_aii_nji_nki_elem); ++ij)
          {
            auto d_ALU_ji = d_ALU + *d_aji + local_tid;
            *d_ALU_ji *= d_Aii_inverse;
            ++d_aji;
          }
          for (std::size_t ik = 0; ik < std::get<2>(d_aii_nji_nki_elem); ++ik)
          {
            const std::size_t d_aik_njk_first = std::get<0>(*d_aik_njk);
            const std::size_t d_aik_njk_second = std::get<1>(*d_aik_njk);
            for (std::size_t ijk = 0; ijk < d_aik_njk_second; ++ijk)
            {
              auto d_ALU_first = d_ALU + d_ajk_aji->first + local_tid;
              auto d_ALU_second = d_ALU + d_ajk_aji->second + local_tid;
              auto d_ALU_aik = d_ALU + d_aik_njk_first + local_tid;
              *d_ALU_first -= *d_ALU_second * *d_ALU_aik;
              ++d_ajk_aji;
            }
            ++d_aik_njk;
          }
        }
      }
    }  // end of CUDA kernel

    /// This is the function that will copy the constant data
    ///   members of class "CudaDoolittleLuDecomposition" to the device
    LuDecomposeMozartInPlaceParam CopyConstData(LuDecomposeMozartInPlaceParam& hoststruct)
    {
      /// Calculate the memory space of each constant data member
      size_t aii_nji_nki_bytes = sizeof(std::tuple<std::size_t, std::size_t, std::size_t>) * hoststruct.aii_nji_nki_size_;
      size_t aji_bytes = sizeof(std::size_t) * hoststruct.aji_size_;
      size_t aik_njk_bytes = sizeof(std::pair<std::size_t, std::size_t>) * hoststruct.aik_njk_size_;
      size_t ajk_aji_bytes = sizeof(std::pair<std::size_t, std::size_t>) * hoststruct.ajk_aji_size_;

      /// Create a struct whose members contain the addresses in the device memory.
      LuDecomposeParam devstruct;
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.aii_nji_nki_), aii_nji_nki_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(&(devstruct.aji_), aji_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.aik_njk_), aik_njk_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");
      CHECK_CUDA_ERROR(
          hipMallocAsync(
              &(devstruct.ajk_aji_), ajk_aji_bytes, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMalloc");

      /// Copy the data from host to device
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.aii_nji_nki_,
              hoststruct.aii_nji_nki_,
              aii_nji_nki_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.aji_,
              hoststruct.aji_,
              aji_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.aik_njk_,
              hoststruct.aik_njk_,
              aik_njk_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");
      CHECK_CUDA_ERROR(
          hipMemcpyAsync(
              devstruct.ajk_aji_,
              hoststruct.ajk_aji_,
              ajk_aji_bytes,
              hipMemcpyHostToDevice,
              micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
          "hipMemcpy");

      /// Copy the other data members from host to device
      devstruct.aii_nji_nki_size_ = hoststruct.aii_nji_nki_size_;
      devstruct.aji_size_ = hoststruct.aji_size_;
      devstruct.aik_njk_size_ = hoststruct.aik_njk_size_;
      devstruct.ajk_aji_size_ = hoststruct.ajk_aji_size_;
      devstruct.number_of_non_zeros_ = hoststruct.number_of_non_zeros_;

      return devstruct;
    }

    /// This is the function that will delete the constant data
    ///   members of class "CudaDoolittleLuDecomposition" on the device
    void FreeConstData(LuDecomposeMozartInPlaceParam& devstruct)
    {
      if (devstruct.aii_nji_nki_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.aii_nji_nki_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)),
            "hipFree");
      if (devstruct.aji_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.aji_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
      if (devstruct.aik_njk_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.aik_njk_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
      if (devstruct.ajk_aji_ != nullptr)
        CHECK_CUDA_ERROR(
            hipFreeAsync(devstruct.ajk_aji_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)), "hipFree");
    }

    void DecomposeKernelDriver(CudaMatrixParam& ALU_param, const LuDecomposeParam& devstruct)
    {
      // Launch the CUDA kernel for LU decomposition
      std::size_t number_of_blocks = (ALU_param.number_of_grid_cells_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      DecomposeKernel<<<number_of_blocks, BLOCK_SIZE, 0, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)>>>(
          ALU_param, devstruct);
    }  // end of DecomposeKernelDriver
  }  // end of namespace cuda
}  // end of namespace micm
