#include "hip/hip_runtime.h"
// Copyright (C) 2023 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0

#include <chrono>
#include <iostream>
#include <micm/util/cuda_matrix_param.hpp>
namespace micm
{
  namespace cuda
  {
    // flipped memory layout
    __global__ void AddForcingTermsKernel(
        double* rate_constants,
        double* state_variables,
        double* forcing,
        size_t n_grids,
        size_t n_reactions,
        size_t n_species,
        size_t* number_of_reactants_,
        size_t* reactant_ids_,
        size_t* number_of_products_,
        size_t* product_ids_,
        double* yields_)
    {
      // define thread index
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      size_t react_id_offset, prod_id_offset, yield_offset;

      if (tid < n_grids)
      {
        react_id_offset = 0;
        prod_id_offset = 0;
        yield_offset = 0;
        for (std::size_t i_rxn = 0; i_rxn < n_reactions; ++i_rxn)
        {
          double rate = rate_constants[i_rxn * n_grids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react)
            rate *= state_variables[reactant_ids_[react_id_offset + i_react] * n_grids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants_[i_rxn]; ++i_react)
          {
            forcing[reactant_ids_[react_id_offset + i_react] * n_grids + tid] -= rate;
          }
          for (std::size_t i_prod = 0; i_prod < number_of_products_[i_rxn]; ++i_prod)
          {
            size_t index = product_ids_[prod_id_offset + i_prod] * n_grids + tid;
            forcing[index] += yields_[yield_offset + i_prod] * rate;
          }
          react_id_offset += number_of_reactants_[i_rxn];
          prod_id_offset += number_of_products_[i_rxn];
          yield_offset += number_of_products_[i_rxn];
        }  // for loop over number of reactions
      }    // if check for valid CUDA threads
    }      // end of AddForcingTerms_kernel

    __global__ void AddJacobianTermsKernel(
        double* rate_constants,
        double* state_variables,
        size_t n_grids,
        size_t n_reactions,
        double* jacobian,
        size_t* number_of_reactants,
        size_t* reactant_ids,
        size_t* number_of_products,
        double* yields,
        size_t* jacobian_flat_ids)
    {
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      size_t react_ids_offset = 0;
      size_t yields_offset = 0;
      size_t flat_id_offset = 0;
      if (tid < n_grids)
      {
        // loop over reactions in a grid
        for (size_t i_rxn = 0; i_rxn < n_reactions; ++i_rxn)
        {
          // loop over reactants in a reaction
          for (size_t i_ind = 0; i_ind < number_of_reactants[i_rxn]; ++i_ind)
          {
            double d_rate_d_ind = rate_constants[i_rxn * n_grids + tid];
            for (size_t i_react = 0; i_react < number_of_reactants[i_rxn]; ++i_react)
            {
              if (i_react != i_ind)
              {
                d_rate_d_ind *= state_variables[reactant_ids[react_ids_offset + i_react] * n_grids + tid];
              }
            }
            for (size_t i_dep = 0; i_dep < number_of_reactants[i_rxn]; ++i_dep)
            {
              size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid;
              jacobian[jacobian_idx] -= d_rate_d_ind;
              flat_id_offset++;
            }
            for (size_t i_dep = 0; i_dep < number_of_products[i_rxn]; ++i_dep)
            {
              size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid;
              jacobian[jacobian_idx] += yields[yields_offset + i_dep] * d_rate_d_ind;
              flat_id_offset++;
            }
          }  // loop over reactants in a reaction
          react_ids_offset += number_of_reactants[i_rxn];
          yields_offset += number_of_products[i_rxn];
        }  // loop over reactions in a grid
      }    // check valid tid
    }      // end of AddJacobianTerms_kernel

    std::chrono::nanoseconds AddJacobianTermsKernelDriver(
        micm::CUDAMatrixParam& matrixParam,
        // const double* rate_constants,
        // const double* state_variables,
        // size_t n_grids,
        // size_t n_reactions,
        // size_t n_species,
        // double* jacobian,
        // size_t jacobian_size,
        const size_t* number_of_reactants,
        const size_t* reactant_ids,
        size_t reactant_ids_size,
        const size_t* number_of_products,
        const double* yields,
        size_t yields_size,
        const size_t* jacobian_flat_ids,
        size_t jacobian_flat_ids_size)
    {
      // create device pointers
      double* d_rate_constants;
      double* d_state_variables;
      double* d_jacobian;
      size_t* d_number_of_reactants;
      size_t* d_reactant_ids;
      size_t* d_number_of_products;
      double* d_yields;
      size_t* d_jacobian_flat_ids;

      // allocate device memory
      hipMalloc(&d_rate_constants, sizeof(double) * matrixParam.n_grids_ * matrixParam.n_reactions_);
      hipMalloc(&d_state_variables, sizeof(double) * matrixParam.n_grids_ * matrixParam.n_species_);
      hipMalloc(&d_jacobian, sizeof(double) * matrixParam.jacobian_size_);
      hipMalloc(&d_number_of_reactants, sizeof(size_t) * matrixParam.n_reactions_);
      hipMalloc(&d_reactant_ids, sizeof(size_t) * reactant_ids_size);
      hipMalloc(&d_number_of_products, sizeof(size_t) * matrixParam.n_reactions_);
      hipMalloc(&d_yields, sizeof(double) * yields_size);
      hipMalloc(&d_jacobian_flat_ids, sizeof(size_t) * jacobian_flat_ids_size);

      // transfer data from host to device
      hipMemcpy(d_rate_constants, matrixParam.rate_constants_, sizeof(double) * matrixParam.n_grids_ * matrixParam.n_reactions_, hipMemcpyHostToDevice);
      hipMemcpy(d_state_variables, matrixParam.state_variables_, sizeof(double) * matrixParam.n_grids_ * matrixParam.n_species_, hipMemcpyHostToDevice);
      hipMemcpy(d_jacobian, matrixParam.jacobian_, sizeof(double) * matrixParam.jacobian_size_, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_reactants, number_of_reactants, sizeof(size_t) * matrixParam.n_reactions_, hipMemcpyHostToDevice);
      hipMemcpy(d_reactant_ids, reactant_ids, sizeof(size_t) * reactant_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_products, number_of_products, sizeof(size_t) * matrixParam.n_reactions_, hipMemcpyHostToDevice);
      hipMemcpy(d_yields, yields, sizeof(double) * yields_size, hipMemcpyHostToDevice);
      hipMemcpy(d_jacobian_flat_ids, jacobian_flat_ids, sizeof(size_t) * jacobian_flat_ids_size, hipMemcpyHostToDevice);

      // setup kernel
      size_t threads_per_block = 320;
      size_t total_blocks = (matrixParam.n_grids_ + threads_per_block - 1) / threads_per_block;

      size_t n_reactions = matrixParam.n_reactions_; 
      size_t n_grids = matrixParam.n_grids_; 
      // launch kernel and measure time performance
      auto startTime = std::chrono::high_resolution_clock::now();
      AddJacobianTermsKernel<<<total_blocks, threads_per_block>>>(
          d_rate_constants,
          d_state_variables,
          n_grids,
          n_reactions,
          d_jacobian,
          d_number_of_reactants,
          d_reactant_ids,
          d_number_of_products,
          d_yields,
          d_jacobian_flat_ids);
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      hipMemcpy(matrixParam.jacobian_, d_jacobian, sizeof(double) * matrixParam.jacobian_size_, hipMemcpyDeviceToHost);
      // clean up
      hipFree(d_rate_constants);
      hipFree(d_state_variables);
      hipFree(d_jacobian);
      hipFree(d_number_of_reactants);
      hipFree(d_reactant_ids);
      hipFree(d_number_of_products);
      hipFree(d_yields);
      hipFree(d_jacobian_flat_ids);
      return kernel_duration;
    }  // end of AddJacobian_kernelSetup

    std::chrono::nanoseconds AddForcingTermsKernelDriver(
        micm::CUDAMatrixParam& matrixParam,
        const size_t* number_of_reactants,
        const size_t* reactant_ids,
        size_t reactant_ids_size,
        const size_t* number_of_products,
        const size_t* product_ids,
        size_t product_ids_size,
        const double* yields,
        size_t yields_size)
    {
      // device pointer to vectorss
      double* d_rate_constants;
      double* d_state_variables;
      double* d_forcing;
      double* d_yields_;
      size_t* d_number_of_reactants_;
      size_t* d_reactant_ids_;
      size_t* d_number_of_products_;
      size_t* d_product_ids_;

      // allocate device memory
      
      hipMalloc(&d_rate_constants, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_reactions_));
      hipMalloc(&d_state_variables, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_species_));
      hipMalloc(&d_forcing, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_species_));
      hipMalloc(&d_number_of_reactants_, sizeof(size_t) * matrixParam.n_reactions_);
      hipMalloc(&d_reactant_ids_, sizeof(size_t) * reactant_ids_size);
      hipMalloc(&d_number_of_products_, sizeof(size_t) * matrixParam.n_reactions_);
      hipMalloc(&d_product_ids_, sizeof(size_t) * product_ids_size);
      hipMalloc(&d_yields_, sizeof(double) * yields_size);

      // copy data from host memory to device memory
      hipMemcpy(d_rate_constants, matrixParam.rate_constants_, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_reactions_), hipMemcpyHostToDevice);
      hipMemcpy(d_state_variables, matrixParam.state_variables_, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_species_), hipMemcpyHostToDevice);
      hipMemcpy(d_forcing, matrixParam.forcing_, sizeof(double) * (matrixParam.n_grids_ * matrixParam.n_species_), hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_reactants_, number_of_reactants, sizeof(size_t) * matrixParam.n_reactions_, hipMemcpyHostToDevice);
      hipMemcpy(d_reactant_ids_, reactant_ids, sizeof(size_t) * reactant_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_products_, number_of_products, sizeof(size_t) * matrixParam.n_reactions_, hipMemcpyHostToDevice);
      hipMemcpy(d_product_ids_, product_ids, sizeof(size_t) * product_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_yields_, yields, sizeof(double) * yields_size, hipMemcpyHostToDevice);

      // total thread count == number of grid cells
      int block_size = 320;
      int num_block = (matrixParam.n_grids_ + block_size - 1) / block_size;

      size_t n_grids = matrixParam.n_grids_; 
      size_t n_reactions = matrixParam.n_reactions_; 
      size_t n_species = matrixParam.n_species_; 
      // launch kernel and measure time performance
      auto startTime = std::chrono::high_resolution_clock::now();
      AddForcingTermsKernel<<<num_block, block_size>>>(
          d_rate_constants,
          d_state_variables,
          d_forcing,
          n_grids,
          n_reactions,
          n_species,
          d_number_of_reactants_,
          d_reactant_ids_,
          d_number_of_products_,
          d_product_ids_,
          d_yields_);
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      // copy data from device memory to host memory
      hipMemcpy(matrixParam.forcing_, d_forcing, sizeof(double) * (n_grids * n_species), hipMemcpyDeviceToHost);

      // clean up
      hipFree(d_rate_constants);
      hipFree(d_state_variables);
      hipFree(d_forcing);
      hipFree(d_number_of_reactants_);
      hipFree(d_reactant_ids_);
      hipFree(d_number_of_products_);
      hipFree(d_product_ids_);
      hipFree(d_yields_);
      return kernel_duration;
    }  // end of AddForcingTerms_kernelSetup
  }    // namespace cuda
}  // namespace micm
