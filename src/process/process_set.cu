#include "hip/hip_runtime.h"
// Copyright (C) 2023 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0

#include <chrono>
#include <iostream>
#include <micm/util/cuda_param.hpp>

//device pointers passing to AddForcingTermsKernel()
typedef struct forcingDevice{
  double* rate_constants; 
  double* state_variables; 
  double* forcing; 
  size_t* number_of_reactants; 
  size_t* reactant_ids; 
  size_t* number_of_products; 
  size_t* product_ids; 
  double* yields; 
};
//device pointers passing to AddJacobianTermsKernel() 
typedef struct jacobianDevice{
  double* rate_constants; 
  double* state_variables; 
  double* jacobian;
  size_t* number_of_reactants; 
  size_t* reactant_ids; 
  size_t* number_of_products; 
  double* yields; 
  size_t* jacobian_flat_ids; 
};
const size_t BLOCK_SIZE = 320;

namespace micm
{
  namespace cuda
  {
    // flipped memory layout
    __global__ void AddForcingTermsKernel(
        forcingDevice* device, 
        size_t n_grids,
        size_t n_reactions,
        size_t n_species)
       
    {
      // define thread index
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      size_t react_id_offset, prod_id_offset, yield_offset;
      double* forcing = device->forcing; 
      size_t* number_of_reactants = device->number_of_reactants; 
      size_t* reactant_ids = device->reactant_ids; 
      size_t* number_of_products = device->number_of_products; 
      if (tid < n_grids)
      {
        react_id_offset = 0;
        prod_id_offset = 0;
        yield_offset = 0;
        for (std::size_t i_rxn = 0; i_rxn < n_reactions; ++i_rxn)
        {
          double rate = device->rate_constants[i_rxn * n_grids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants[i_rxn]; ++i_react)
            rate *= device->state_variables[reactant_ids[react_id_offset + i_react] * n_grids + tid];
          for (std::size_t i_react = 0; i_react < number_of_reactants[i_rxn]; ++i_react)
          {
            forcing[reactant_ids[react_id_offset + i_react] * n_grids + tid] -= rate;
          }
          for (std::size_t i_prod = 0; i_prod < number_of_products[i_rxn]; ++i_prod)
          {
            size_t index = device->product_ids[prod_id_offset + i_prod] * n_grids + tid;
            forcing[index] += device->yields[yield_offset + i_prod] * rate;
          }
          react_id_offset += number_of_reactants[i_rxn];
          prod_id_offset += number_of_products[i_rxn];
          yield_offset += number_of_products[i_rxn];
        }  // for loop over number of reactions
      }    // if check for valid CUDA threads
    }      // end of AddForcingTerms_kernel

    __global__ void AddJacobianTermsKernel(
        jacobianDevice* device,
        size_t n_grids,
        size_t n_reactions)
    {
      size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
      size_t react_ids_offset = 0;
      size_t yields_offset = 0;
      size_t flat_id_offset = 0;
      size_t* number_of_reactants = device->number_of_reactants; 
      size_t* jacobian_flat_ids = device->jacobian_flat_ids; 
      size_t* number_of_products = device->number_of_products; 
      double* jacobian = device->jacobian; 
      
      if (tid < n_grids)
      {
        // loop over reactions in a grid
        for (size_t i_rxn = 0; i_rxn < n_reactions; ++i_rxn)
        {
          // loop over reactants in a reaction
          for (size_t i_ind = 0; i_ind < number_of_reactants[i_rxn]; ++i_ind)
          {
            double d_rate_d_ind = device->rate_constants[i_rxn * n_grids + tid];
            for (size_t i_react = 0; i_react < number_of_reactants[i_rxn]; ++i_react)
            {
              if (i_react != i_ind)
              {
                d_rate_d_ind *= device->state_variables[device->reactant_ids[react_ids_offset + i_react] * n_grids + tid];
              }
            }
            for (size_t i_dep = 0; i_dep < number_of_reactants[i_rxn]; ++i_dep)
            {
              size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid;
              jacobian[jacobian_idx] -= d_rate_d_ind;
              flat_id_offset++;
            }
            for (size_t i_dep = 0; i_dep < number_of_products[i_rxn]; ++i_dep)
            {
              size_t jacobian_idx = jacobian_flat_ids[flat_id_offset] + tid;
              jacobian[jacobian_idx] += device->yields[yields_offset + i_dep] * d_rate_d_ind;
              flat_id_offset++;
            }
          }  // loop over reactants in a reaction
          react_ids_offset += number_of_reactants[i_rxn];
          yields_offset += number_of_products[i_rxn];
        }  // loop over reactions in a grid
      }    // check valid tid
    }      // end of AddJacobianTerms_kernel

    std::chrono::nanoseconds AddJacobianTermsKernelDriver(
        CUDAMatrixParam& matrixParam,
        CUDASparseMatrixParam& sparseMatrix, 
        CUDAProcessSetParam& processSet)
    {
      // create device pointers
      double* d_rate_constants;
      double* d_state_variables;
      double* d_jacobian;
      size_t* d_number_of_reactants;
      size_t* d_reactant_ids;
      size_t* d_number_of_products;
      double* d_yields;
      size_t* d_jacobian_flat_ids;
      jacobianDevice* device; 

      // allocate device memory
      hipMalloc(&d_rate_constants, sizeof(double) * matrixParam.n_grids * matrixParam.n_reactions);
      hipMalloc(&d_state_variables, sizeof(double) * matrixParam.n_grids * matrixParam.n_species);
      hipMalloc(&d_jacobian, sizeof(double) * sparseMatrix.jacobian_size);
      hipMalloc(&d_number_of_reactants, sizeof(size_t) * matrixParam.n_reactions);
      hipMalloc(&d_reactant_ids, sizeof(size_t) * processSet.reactant_ids_size);
      hipMalloc(&d_number_of_products, sizeof(size_t) * matrixParam.n_reactions);
      hipMalloc(&d_yields, sizeof(double) * processSet.yields_size);
      hipMalloc(&d_jacobian_flat_ids, sizeof(size_t) * processSet.jacobian_flat_ids_size);
      hipMalloc(&device, sizeof(jacobianDevice)); 
      

      // transfer data from host to device
      hipMemcpy(d_rate_constants, matrixParam.rate_constants, sizeof(double) * matrixParam.n_grids * matrixParam.n_reactions, hipMemcpyHostToDevice);
      hipMemcpy(d_state_variables, matrixParam.state_variables, sizeof(double) * matrixParam.n_grids * matrixParam.n_species, hipMemcpyHostToDevice);
      hipMemcpy(d_jacobian, sparseMatrix.jacobian, sizeof(double) * sparseMatrix.jacobian_size, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_reactants, processSet.number_of_reactants, sizeof(size_t) * matrixParam.n_reactions, hipMemcpyHostToDevice);
      hipMemcpy(d_reactant_ids, processSet.reactant_ids, sizeof(size_t) * processSet.reactant_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_products, processSet.number_of_products, sizeof(size_t) * matrixParam.n_reactions, hipMemcpyHostToDevice);
      hipMemcpy(d_yields, processSet.yields, sizeof(double) * processSet.yields_size, hipMemcpyHostToDevice);
      hipMemcpy(d_jacobian_flat_ids, processSet.jacobian_flat_ids, sizeof(size_t) * processSet.jacobian_flat_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(&(device->rate_constants), &d_rate_constants, sizeof(double*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->state_variables), &d_state_variables, sizeof(double*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->jacobian), &d_jacobian, sizeof(double*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->number_of_reactants), &d_number_of_reactants, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->reactant_ids), &d_reactant_ids, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->number_of_products), &d_number_of_products, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->yields), &d_yields, sizeof(double*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->jacobian_flat_ids), &d_jacobian_flat_ids, sizeof(size_t*), hipMemcpyHostToDevice); 

      
      // setup kernel
      size_t total_blocks = (matrixParam.n_grids + BLOCK_SIZE - 1) / BLOCK_SIZE;

      size_t n_reactions = matrixParam.n_reactions; 
      size_t n_grids = matrixParam.n_grids; 
      // launch kernel and measure time performance
      auto startTime = std::chrono::high_resolution_clock::now();
      AddJacobianTermsKernel<<<total_blocks, BLOCK_SIZE>>>(
          device,
          n_grids,
          n_reactions);
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      hipMemcpy(sparseMatrix.jacobian, d_jacobian, sizeof(double) * sparseMatrix.jacobian_size, hipMemcpyDeviceToHost);
      // clean up
      hipFree(d_rate_constants);
      hipFree(d_state_variables);
      hipFree(d_jacobian);
      hipFree(d_number_of_reactants);
      hipFree(d_reactant_ids);
      hipFree(d_number_of_products);
      hipFree(d_yields);
      hipFree(d_jacobian_flat_ids);
      hipFree(device); 
      return kernel_duration;
    }  // end of AddJacobian_kernelSetup

    std::chrono::nanoseconds AddForcingTermsKernelDriver(
        CUDAMatrixParam& matrixParam,
        CUDAProcessSetParam& processSet)
    {
      // device pointer to vectorss
      double* d_rate_constants;
      double* d_state_variables;
      double* d_forcing;
      double* d_yields;
      size_t* d_number_of_reactants;
      size_t* d_reactant_ids;
      size_t* d_number_of_products;
      size_t* d_product_ids;
      forcingDevice* device; 

      // allocate device memory
      hipMalloc(&d_rate_constants, sizeof(double) * (matrixParam.n_grids * matrixParam.n_reactions));
      hipMalloc(&d_state_variables, sizeof(double) * (matrixParam.n_grids * matrixParam.n_species));
      hipMalloc(&d_forcing, sizeof(double) * (matrixParam.n_grids * matrixParam.n_species));
      hipMalloc(&d_number_of_reactants, sizeof(size_t) * matrixParam.n_reactions);
      hipMalloc(&d_reactant_ids, sizeof(size_t) * processSet.reactant_ids_size);
      hipMalloc(&d_number_of_products, sizeof(size_t) * matrixParam.n_reactions);
      hipMalloc(&d_product_ids, sizeof(size_t) * processSet.product_ids_size);
      hipMalloc(&d_yields, sizeof(double) * processSet.yields_size);
      hipMalloc(&device, sizeof(forcingDevice)); 

      // copy data from host memory to device memory
      hipMemcpy(d_rate_constants, matrixParam.rate_constants, sizeof(double) * (matrixParam.n_grids * matrixParam.n_reactions), hipMemcpyHostToDevice);
      hipMemcpy(d_state_variables, matrixParam.state_variables, sizeof(double) * (matrixParam.n_grids * matrixParam.n_species), hipMemcpyHostToDevice);
      hipMemcpy(d_forcing, matrixParam.forcing, sizeof(double) * (matrixParam.n_grids * matrixParam.n_species), hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_reactants, processSet.number_of_reactants, sizeof(size_t) * matrixParam.n_reactions, hipMemcpyHostToDevice);
      hipMemcpy(d_reactant_ids, processSet.reactant_ids, sizeof(size_t) * processSet.reactant_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_number_of_products, processSet.number_of_products, sizeof(size_t) * matrixParam.n_reactions, hipMemcpyHostToDevice);
      hipMemcpy(d_product_ids, processSet.product_ids, sizeof(size_t) * processSet.product_ids_size, hipMemcpyHostToDevice);
      hipMemcpy(d_yields, processSet.yields, sizeof(double) * processSet.yields_size, hipMemcpyHostToDevice);
      hipMemcpy(&(device->rate_constants), &d_rate_constants, sizeof(double*),hipMemcpyHostToDevice); 
      hipMemcpy(&(device->state_variables), &d_state_variables, sizeof(double*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->forcing), &d_forcing, sizeof(double*), hipMemcpyHostToDevice);   
      hipMemcpy(&(device->number_of_reactants), &d_number_of_reactants, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->reactant_ids), &d_reactant_ids, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->number_of_products), &d_number_of_products, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->product_ids), &d_product_ids, sizeof(size_t*), hipMemcpyHostToDevice); 
      hipMemcpy(&(device->yields), &d_yields, sizeof(double*), hipMemcpyHostToDevice); 

      // total thread count == number of grid cells
     
      int num_block = (matrixParam.n_grids + BLOCK_SIZE - 1) / BLOCK_SIZE;

      size_t n_grids = matrixParam.n_grids; 
      size_t n_reactions = matrixParam.n_reactions; 
      size_t n_species = matrixParam.n_species; 
      
      // launch kernel and measure time performance
      auto startTime = std::chrono::high_resolution_clock::now();
      AddForcingTermsKernel<<<num_block, BLOCK_SIZE>>>(
          device,
          n_grids,
          n_reactions,
          n_species);
      hipDeviceSynchronize();
      auto endTime = std::chrono::high_resolution_clock::now();
      auto kernel_duration = std::chrono::duration_cast<std::chrono::nanoseconds>(endTime - startTime);

      // copy data from device memory to host memory
      hipMemcpy(matrixParam.forcing, d_forcing, sizeof(double) * (n_grids * n_species), hipMemcpyDeviceToHost);

      // clean up
      hipFree(d_rate_constants);
      hipFree(d_state_variables);
      hipFree(d_forcing);
      hipFree(d_number_of_reactants);
      hipFree(d_reactant_ids);
      hipFree(d_number_of_products);
      hipFree(d_product_ids);
      hipFree(d_yields);
      hipFree(device); 
      return kernel_duration;
    }  // end of AddForcingTerms_kernelSetup
  }    // namespace cuda
}  // namespace micm
