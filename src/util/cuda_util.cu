// Copyright (C) 2023-2024 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <micm/cuda/util/cuda_util.cuh>
#include <micm/util/internal_error.hpp>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <map>
#include <memory>
#include <mutex>

namespace micm
{
  namespace cuda
  {
    void CheckCudaError(hipError_t err, const char* file, int line, std::string str)
    {
      if (err != hipSuccess)
      {
        std::string msg = std::string(hipGetErrorString(err)) + " : " + str;
        ThrowInternalError(MicmInternalErrc::Cuda, file, line, msg.c_str());
      }
    }

    void CheckCublasError(hipblasStatus_t err, const char* file, int line, std::string str)
    {
      if (err != HIPBLAS_STATUS_SUCCESS)
      {
        std::string msg = std::to_string(err) + " : " + str;
        ThrowInternalError(MicmInternalErrc::Cublas, file, line, msg.c_str());
      }
    }

    // Define a functor for the hipblasHandle_t unique pointer deleter
    struct CublasHandleDeleter
    {
      void operator()(hipblasHandle_t* handle) const
      {
        if (handle != nullptr)
        {
          CHECK_CUBLAS_ERROR(hipblasDestroy(*handle), "CUBLAS finalization failed");
          delete handle;
        }
      }
    };

    // Define the smart pointer type using the functor for the custom deleter
    using CublasHandlePtr = std::unique_ptr<hipblasHandle_t, CublasHandleDeleter>;

    // Create a cublas handle and return a unique pointer to it
    CublasHandlePtr CreateCublasHandle()
    {
      hipblasHandle_t* handle = new hipblasHandle_t;
      CHECK_CUBLAS_ERROR(hipblasCreate(handle), "CUBLAS initialization failed...");
      return CublasHandlePtr(handle, CublasHandleDeleter());
    }

    hipblasHandle_t& GetCublasHandle()
    {
      static std::map<int, CublasHandlePtr> cublas_handles_map;
      static std::mutex mutex;
      int device_id;
      CHECK_CUDA_ERROR(hipGetDevice(&device_id), "Failed to get device ID...");
      std::lock_guard<std::mutex> lock(mutex);  // lock the mutex and generate a new cublas handle below
      if (auto search = cublas_handles_map.find(device_id); search == cublas_handles_map.end())
      {
        cublas_handles_map[device_id] = std::move(CreateCublasHandle());
      }
      return *cublas_handles_map[device_id];
    }
  }  // namespace cuda
}  // namespace micm
