// Copyright (C) 2023-2024 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <micm/cuda/util/cuda_util.cuh>
#include <micm/util/internal_error.hpp>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <map>
#include <memory>
#include <mutex>

namespace micm
{
  namespace cuda
  {
    void CheckCudaError(hipError_t err, const char* file, int line, std::string str)
    {
      if (err != hipSuccess)
      {
        std::string msg = std::string(hipGetErrorString(err)) + " : " + str;
        ThrowInternalError(MicmInternalErrc::Cuda, file, line, msg.c_str());
      }
    }

    void CheckCublasError(hipblasStatus_t err, const char* file, int line, std::string str)
    {
      if (err != HIPBLAS_STATUS_SUCCESS)
      {
        std::string msg = std::to_string(err) + " : " + str;
        ThrowInternalError(MicmInternalErrc::Cublas, file, line, msg.c_str());
      }
    }

    /*
       The following functions are used to create and manage cublas handles
    */

    // Define a functor for the hipblasHandle_t unique pointer deleter
    struct CublasHandleDeleter
    {
      void operator()(hipblasHandle_t* handle) const
      {
        if (handle != nullptr)
        {
          CHECK_CUBLAS_ERROR(hipblasDestroy(*handle), "CUBLAS finalization failed");
          delete handle;
        }
      }
    };

    // Define the smart pointer type using the functor for the custom deleter
    using CublasHandlePtr = std::unique_ptr<hipblasHandle_t, CublasHandleDeleter>;

    // Create a cublas handle and return a unique pointer to it
    CublasHandlePtr CreateCublasHandle()
    {
      hipblasHandle_t* handle = new hipblasHandle_t;
      CHECK_CUBLAS_ERROR(hipblasCreate(handle), "CUBLAS initialization failed...");
      return CublasHandlePtr(handle, CublasHandleDeleter());
    }

    // Get the cublas handle for the current device
    hipblasHandle_t& GetCublasHandle()
    {
      static std::map<int, CublasHandlePtr> cublas_handles_map;
      static std::mutex mutex;
      int device_id;
      CHECK_CUDA_ERROR(hipGetDevice(&device_id), "Failed to get device ID...");
      std::lock_guard<std::mutex> lock(mutex);  // lock the mutex and generate a new cublas handle below
      if (auto search = cublas_handles_map.find(device_id); search == cublas_handles_map.end())
      {
        cublas_handles_map[device_id] = std::move(CreateCublasHandle());
      }
      return *cublas_handles_map[device_id];
    }

    /*
        The following functions are used to create and manage cuda streams
    */

    // Define a functor for the cudaStream unique pointer deleter
    struct CudaStreamDeleter
    {
      void operator()(hipStream_t* hip_stream) const
      {
        if (hip_stream != nullptr)
        {
          hipStreamSynchronize(*hip_stream);
          CHECK_CUDA_ERROR(hipStreamDestroy(*hip_stream), "CUDA stream finalization failed");
          delete hip_stream;
        }
      }
    };

    // Define the smart pointer type using the functor for the custom deleter
    using CudaStreamPtr = std::unique_ptr<hipStream_t, CudaStreamDeleter>;

    // Create a CUDA stream and return a unique pointer to it
    CudaStreamPtr CreateCudaStream()
    {
      hipStream_t* hip_stream = new hipStream_t;
      CHECK_CUDA_ERROR(hipStreamCreate(hip_stream), "CUDA stream initialization failed...");
      return CudaStreamPtr(hip_stream, CudaStreamDeleter());
    }

    // Get the CUDA stream given a stream ID
    hipStream_t& GetCudaStream(std::size_t stream_id)
    {
      static std::map<int, CudaStreamPtr> cuda_streams_map;
      if (auto search = cuda_streams_map.find(stream_id); search == cuda_streams_map.end())
      {
        cuda_streams_map[stream_id] = std::move(CreateCudaStream());
      }
      return *cuda_streams_map[stream_id];
    } 
  }  // namespace cuda
}  // namespace micm
