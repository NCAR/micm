#include <micm/util/cuda_vector_matrix.cuh>
#include <vector>

namespace micm
{
  namespace cuda
  {
    int MallocVector(CudaVectorMatrixParam& param, std::size_t num_elements)
    {
      param.num_elements_ = num_elements;
      return hipMalloc(&(param.d_data_), sizeof(double) * num_elements);
    }
    int FreeVector(CudaVectorMatrixParam& param)
    {
      return hipFree(param.d_data_);
    }
    int CopyToDevice(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.num_elements_, hipMemcpyHostToDevice);
    }
    int CopyToHost(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.num_elements_, hipMemcpyDeviceToHost);
    }
    int CopyToDeviceFromDevice(CudaVectorMatrixParam& vectorMatrixDest, const CudaVectorMatrixParam& vectorMatrixSrc)
    {
      return hipMemcpy(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(double) * vectorMatrixSrc.num_elements_,
          hipMemcpyDeviceToDevice);
    }
  }  // namespace cuda
}  // namespace micm
