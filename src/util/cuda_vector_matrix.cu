
#include <hip/hip_runtime.h>
#include <vector>
//#include <micm/util/cuda_vector_matrix.cuh>

namespace micm
{
  namespace cuda
  {
    int malloc_vector(double *d_data, std::size_t num_elements)
    {
      return hipMalloc(&d_data, sizeof(double) * num_elements);
    }
    int free_vector(double *d_data)
    {
      return hipFree(d_data);
    }
    int copy_to_device(double *d_data, const double *h_data, std::size_t num_elements)
    {
      return hipMemcpy(d_data, h_data, sizeof(double) * num_elements, hipMemcpyHostToDevice);
    }
    int copy_to_host(double* d_data, double *h_data, std::size_t num_elements)
    {
      return hipMemcpy(h_data, d_data, sizeof(double) * num_elements, hipMemcpyDeviceToHost);
    }
  }
}
