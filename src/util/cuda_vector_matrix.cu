#include <vector>
#include <micm/util/cuda_vector_matrix.cuh>

namespace micm
{
  namespace cuda
  {
    int MallocVector(CudaVectorMatrixParam& param, std::size_t num_elements)
    {
      param.num_elements_ = num_elements;
      return hipMalloc(&(param.d_data_), sizeof(double) * num_elements);
    }
    int FreeVector(CudaVectorMatrixParam& param)
    {
      return hipFree(param.d_data_);
    }
    int CopyToDevice(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.num_elements_, hipMemcpyHostToDevice);
    }
    int CopyToHost(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.num_elements_, hipMemcpyDeviceToHost);
    }
  }
}
