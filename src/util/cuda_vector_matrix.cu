
#include <hip/hip_runtime.h>
#include <vector>
//#include <micm/util/cuda_vector_matrix.cuh>

namespace micm
{
  namespace cuda
  {
    int MallocVector(double *d_data, std::size_t num_elements)
    {
      return hipMalloc(&d_data, sizeof(double) * num_elements);
    }
    int FreeVector(double *d_data)
    {
      return hipFree(d_data);
    }
    int CopyToDevice(double *d_data, const double *h_data, std::size_t num_elements)
    {
      return hipMemcpy(d_data, h_data, sizeof(double) * num_elements, hipMemcpyHostToDevice);
    }
    int CopyToHost(double* d_data, double *h_data, std::size_t num_elements)
    {
      return hipMemcpy(h_data, d_data, sizeof(double) * num_elements, hipMemcpyDeviceToHost);
    }
  }
}
