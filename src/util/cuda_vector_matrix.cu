#include <hip/hip_runtime.h>

#include <micm/util/cuda_vector_matrix.cuh>
#include <vector>

namespace micm
{
  namespace cuda
  {
    int MallocVector(CudaVectorMatrixParam& param, std::size_t number_of_elements)
    {
      param.number_of_elements_ = number_of_elements;
      return hipMalloc(&(param.d_data_), sizeof(double) * number_of_elements);
    }
    int FreeVector(CudaVectorMatrixParam& param)
    {
      return hipFree(param.d_data_);
    }
    int CopyToDevice(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.number_of_elements_, hipMemcpyHostToDevice);
    }
    int CopyToHost(CudaVectorMatrixParam& param, std::vector<double>& h_data)
    {
      hipDeviceSynchronize();
      return hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.number_of_elements_, hipMemcpyDeviceToHost);
    }
    int CopyToDeviceFromDevice(CudaVectorMatrixParam& vectorMatrixDest, const CudaVectorMatrixParam& vectorMatrixSrc)
    {
      return hipMemcpy(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(double) * vectorMatrixSrc.number_of_elements_,
          hipMemcpyDeviceToDevice);
    }
  }  // namespace cuda
}  // namespace micm
