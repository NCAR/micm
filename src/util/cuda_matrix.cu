#include <hip/hip_runtime.h>

#include <micm/util/cuda_matrix.cuh>
#include <vector>

namespace micm
{
  namespace cuda
  {
    int MallocVector(CudaMatrixParam& param, std::size_t number_of_elements)
    {
      param.number_of_elements_ = number_of_elements;
      return hipMalloc(&(param.d_data_), sizeof(double) * number_of_elements);
    }
    int FreeVector(CudaMatrixParam& param)
    {
      return hipFree(param.d_data_);
    }
    int CopyToDevice(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      return hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.number_of_elements_, hipMemcpyHostToDevice);
    }
    int CopyToHost(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      hipDeviceSynchronize();
      return hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.number_of_elements_, hipMemcpyDeviceToHost);
    }
    int CopyToDeviceFromDevice(CudaMatrixParam& vectorMatrixDest, const CudaMatrixParam& vectorMatrixSrc)
    {
      return hipMemcpy(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(double) * vectorMatrixSrc.number_of_elements_,
          hipMemcpyDeviceToDevice);
    }
  }  // namespace cuda
}  // namespace micm
