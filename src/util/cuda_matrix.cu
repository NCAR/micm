// Copyright (C) 2023-2024 National Center for Atmospheric Research,
//
// SPDX-License-Identifier: Apache-2.0
#pragma once

#include <hip/hip_runtime.h>

#include <iostream>
#include <micm/util/cuda_matrix.cuh>
#include <vector>

namespace micm
{
  namespace cuda
  {
    hipError_t MallocVector(CudaMatrixParam& param, std::size_t number_of_elements)
    {
      param.number_of_elements_ = number_of_elements;
      hipError_t err = hipMalloc(&(param.d_data_), sizeof(double) * number_of_elements);
      return err;
    }

    hipError_t FreeVector(CudaMatrixParam& param)
    {
      if (param.d_data_ == nullptr)
      {
        return hipError_t::hipSuccess;
      }
      hipError_t err = hipFree(param.d_data_);
      param.d_data_ = nullptr;
      return err;
    }

    hipError_t CopyToDevice(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      hipError_t err =
          hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.number_of_elements_, hipMemcpyHostToDevice);
      return err;
    }

    hipError_t CopyToHost(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      hipDeviceSynchronize();
      hipError_t err =
          hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.number_of_elements_, hipMemcpyDeviceToHost);
      return err;
    }

    hipError_t CopyToDeviceFromDevice(CudaMatrixParam& vectorMatrixDest, const CudaMatrixParam& vectorMatrixSrc)
    {
      hipError_t err = hipMemcpy(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(double) * vectorMatrixSrc.number_of_elements_,
          hipMemcpyDeviceToDevice);
      return err;
    }

    void CheckCudaError(hipError_t err, const char* file, int line, std::string str)
    {
      if (err != hipSuccess)
      {
        std::cout << "CUDA error: " << hipGetErrorString(err) << " at " << file << ":" << line << std::endl;
        throw std::runtime_error(str + " failed...");
      }
    }

    void CheckCublasError(hipblasStatus_t err, const char* file, int line, std::string str)
    {
      if (err != HIPBLAS_STATUS_SUCCESS)
      {
        std::cout << "CUBLAS error: " << err << " at " << file << ":" << line << std::endl;
        throw std::runtime_error(str);
      }
    }
  }  // namespace cuda
}  // namespace micm
