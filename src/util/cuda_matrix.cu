#include "hip/hip_runtime.h"
// Copyright (C) 2023-2024 National Center for Atmospheric Research
// SPDX-License-Identifier: Apache-2.0
#include <micm/cuda/util/cuda_matrix.cuh>
#include <micm/cuda/util/cuda_param.hpp>
#include <micm/cuda/util/cuda_util.cuh>
#include <micm/util/internal_error.hpp>

#include <hip/hip_runtime.h>

#include <vector>

namespace micm
{
  namespace cuda
  {
    template<typename T>
    hipError_t MallocVector(CudaMatrixParam& param, std::size_t number_of_elements)
    {
      param.number_of_elements_ = number_of_elements;
      hipError_t err = hipMallocAsync(&(param.d_data_), sizeof(T) * number_of_elements, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      return err;
    }

    hipError_t FreeVector(CudaMatrixParam& param)
    {
      param.number_of_elements_ = 0;
      param.number_of_grid_cells_ = 0;
      if (param.d_data_ == nullptr)
      {
        return hipError_t::hipSuccess;
      }
      hipError_t err = hipFreeAsync(param.d_data_, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      param.d_data_ = nullptr;
      return err;
    }

    template<typename T>
    hipError_t CopyToDevice(CudaMatrixParam& param, std::vector<T>& h_data)
    {
      hipError_t err =
      hipMemcpyAsync(param.d_data_, h_data.data(), sizeof(T) * param.number_of_elements_, hipMemcpyHostToDevice, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      return err;
    }

    template<typename T>
    hipError_t CopyToHost(CudaMatrixParam& param, std::vector<T>& h_data)
    {
      hipError_t err =
        hipMemcpyAsync(h_data.data(), param.d_data_, sizeof(T) * param.number_of_elements_, hipMemcpyDeviceToHost, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      hipStreamSynchronize(micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      return err;
    }

    template<typename T>
    hipError_t CopyToDeviceFromDevice(CudaMatrixParam& vectorMatrixDest, const CudaMatrixParam& vectorMatrixSrc)
    {
      hipError_t err = hipMemcpyAsync(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(T) * vectorMatrixSrc.number_of_elements_,
          hipMemcpyDeviceToDevice,
          micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0));
      return err;
    }

    template<typename T>
    __global__ void FillCudaMatrixKernel(T* d_data, std::size_t number_of_elements, T val)
    {
      std::size_t tid = blockIdx.x * BLOCK_SIZE + threadIdx.x;
      if (tid < number_of_elements)
      {
        d_data[tid] = val;
      }
    }

    template<typename T>
    hipError_t FillCudaMatrix(CudaMatrixParam& param, T val)
    {
      std::size_t number_of_blocks = (param.number_of_elements_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
      FillCudaMatrixKernel<<<number_of_blocks, BLOCK_SIZE, 0, micm::cuda::CudaStreamSingleton::GetInstance().GetCudaStream(0)>>>(param.d_data_, param.number_of_elements_, val);
      hipError_t err = hipGetLastError();
      return err;
    }

    // source code needs the instantiation of the template
    template hipError_t MallocVector<double>(CudaMatrixParam& param, std::size_t number_of_elements);
    template hipError_t MallocVector<int>(CudaMatrixParam& param, std::size_t number_of_elements);
    template hipError_t CopyToDevice<double>(CudaMatrixParam& param, std::vector<double>& h_data);
    template hipError_t CopyToHost<double>(CudaMatrixParam& param, std::vector<double>& h_data);
    template hipError_t CopyToDeviceFromDevice<double>(
        CudaMatrixParam& vectorMatrixDest,
        const CudaMatrixParam& vectorMatrixSrc);
    template hipError_t CopyToDeviceFromDevice<int>(
        CudaMatrixParam& vectorMatrixDest,
        const CudaMatrixParam& vectorMatrixSrc);
    template hipError_t FillCudaMatrix<double>(CudaMatrixParam& param, double val);
  }  // namespace cuda
}  // namespace micm
