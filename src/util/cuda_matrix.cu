/* Copyright (C) 2023-2024 National Center for Atmospheric Research
 *
 * SPDX-License-Identifier: Apache-2.0
 */
#include <micm/util/cuda_matrix.cuh>
#include <micm/util/internal_error.hpp>

#include <hip/hip_runtime.h>

#include <vector>

namespace micm
{
  namespace cuda
  {
    hipError_t MallocVector(CudaMatrixParam& param, std::size_t number_of_elements)
    {
      param.number_of_elements_ = number_of_elements;
      hipError_t err = hipMalloc(&(param.d_data_), sizeof(double) * number_of_elements);
      return err;
    }

    hipError_t FreeVector(CudaMatrixParam& param)
    {
      param.number_of_elements_ = 0;
      param.number_of_grid_cells_ = 0;
      if (param.d_data_ == nullptr)
      {
        return hipError_t::hipSuccess;
      }
      hipError_t err = hipFree(param.d_data_);
      param.d_data_ = nullptr;
      return err;
    }

    hipError_t CopyToDevice(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      hipError_t err =
          hipMemcpy(param.d_data_, h_data.data(), sizeof(double) * param.number_of_elements_, hipMemcpyHostToDevice);
      return err;
    }

    hipError_t CopyToHost(CudaMatrixParam& param, std::vector<double>& h_data)
    {
      hipDeviceSynchronize();
      hipError_t err =
          hipMemcpy(h_data.data(), param.d_data_, sizeof(double) * param.number_of_elements_, hipMemcpyDeviceToHost);
      return err;
    }

    hipError_t CopyToDeviceFromDevice(CudaMatrixParam& vectorMatrixDest, const CudaMatrixParam& vectorMatrixSrc)
    {
      hipError_t err = hipMemcpy(
          vectorMatrixDest.d_data_,
          vectorMatrixSrc.d_data_,
          sizeof(double) * vectorMatrixSrc.number_of_elements_,
          hipMemcpyDeviceToDevice);
      return err;
    }
  }  // namespace cuda
}  // namespace micm
